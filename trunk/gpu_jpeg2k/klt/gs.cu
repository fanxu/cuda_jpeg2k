#include "hip/hip_runtime.h"
/* 
Copyright 2009-2013 Poznan Supercomputing and Networking Center

Authors:
Milosz Ciznicki miloszc@man.poznan.pl

GPU JPEG2K is free software: you can redistribute it and/or modify
it under the terms of the GNU Affero General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

GPU JPEG2K is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU Affero General Public License for more details.

You should have received a copy of the GNU Affero General Public License
along with GPU JPEG2K. If not, see <http://www.gnu.org/licenses/>.
*/
/**
 * @file gs.cu
 *
 * @author Kamil Balwierz
 */

#include <math.h>
#include <time.h>
#include "gs.h"
#include <hipblas.h>


int gram_schmidt(int N, type_data* output, type_data *dinput, type_data *eValues, int J, type_data er) {
	hipblasStatus_t status;
	int j, k;
	type_data *dT = 0;
	status = cublasAlloc(N*N, sizeof (type_data), (void**) &dT);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "! device memory allocation error (dT)\n");
		return EXIT_FAILURE;
	}
	type_data *doutput = 0;
	status = cublasAlloc(N*N, sizeof (type_data), (void**) &doutput);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "! device memory allocation error (doutput)\n");
		return EXIT_FAILURE;
	}
	if (eValues == 0) {
		fprintf(stderr, "! host memory allocation error: T\n");
		return EXIT_FAILURE;
	}
	type_data *dU = 0;
	status = cublasAlloc(N, sizeof (type_data), (void**) &dU);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "! device memory allocation error (dU)\n");
		return EXIT_FAILURE;
	}

	int iter = 0;
	type_data a;
	for (k = 0; k < N; k++) {
		hipblasScopy(N, &dinput[k * N], 1, &dT[k * N], 1);
		a = 0.0;
		for (j = 0; j < J; j++) {
			hipblasSgemv('t', N, N, 1.0f, dinput, N, &dT[k * N], 1, 0.0f, &doutput[k * N], 1);
			if (k > 0) {
				hipblasSgemv('t', N, k, 1.0f, doutput, N, &doutput[k * N], 1, 0.0f, dU, 1);
				hipblasSgemv('n', N, k, -1.0f, doutput, N, dU, 1, 1.0f, &doutput[k * N], 1);
			}
			hipblasSscal(N, 1.0f / hipblasSnrm2(N, &doutput[k * N], 1), &doutput[k * N], 1);
			hipblasSgemv('n', N, N, 1.0f, dinput, N, &doutput[k * N], 1, 0.0f, &dT[k * N], 1);
			if (k > 0) {
				hipblasSgemv('t', N, k, 1.0f, dT, N, &dT[k * N], 1, 0.0, dU, 1);
				hipblasSgemv('n', N, k, -1.0f, dT, N, dU, 1, 1.0f, &dT[k * N], 1);
			}
			eValues[k] = hipblasSnrm2(N, &dT[k * N], 1);
			hipblasSscal(N, 1.0f / eValues[k], &dT[k * N], 1);
//			printf("%f\n", fabs(a - eValues[k]));
			if (fabs(a - eValues[k]) < er * eValues[k]) {
//				printf("iterations %d\n", j);
				break;
			}
			a = eValues[k];
			iter++;
		}
//		iter = 0;
		hipblasSger(N, N, -eValues[k], &dT[k * N], 1, &doutput[k * N], 1, dinput, N);
	}
	printf("iter %d\n", iter);
	for (k = 0; k < N; k++) {
		hipblasSscal(N, eValues[k], &dT[k * N], 1);
	}
//	hipblasSgemm('n', 'n', N, N, N, 1.0, dT, N, doutput, N, 0.0, output, N);
	hipblasGetMatrix(N, N, sizeof (doutput[0]), doutput, N, output, N);
	status = cublasFree(doutput);
	status = cublasFree(dT);
	return EXIT_SUCCESS;
}
