/* 
Copyright 2009-2013 Poznan Supercomputing and Networking Center

Authors:
Milosz Ciznicki miloszc@man.poznan.pl

GPU JPEG2K is free software: you can redistribute it and/or modify
it under the terms of the GNU Affero General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

GPU JPEG2K is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU Affero General Public License for more details.

You should have received a copy of the GNU Affero General Public License
along with GPU JPEG2K. If not, see <http://www.gnu.org/licenses/>.
*/
/**
 * @file memory_management.cu
 *
 * @brief CUDA memory management functions wrappers.
 *
 * This a collection of wrappers for CUDA MM that include debugging information logging and error checking.
 *
 * @author Miłosz Ciżnicki
 * @author Jakub Misiorny <misiorny@man.poznan.pl>
 */
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdint.h>
#include <stdio.h>


#include "cuda_errors.h"
extern "C" {
	#include "memory_management.cuh"
#include "../print_info/print_info.h"
}

void cuda_d_free(void *data)
{
	hipFree(data);
	checkCUDAError("cuda_d_free");
}

void cuda_h_free(void *data) {
	hipHostFree(data);
//	checkCUDAError("cuda_h_free");
}

void cuda_set_device_flags() {
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	hipSetDeviceFlags(hipDeviceMapHost);

	if (!prop.canMapHostMemory) {
		printf("[allocate_mem]: Cannot allocate host-device mapped memory. Exitting!\n");
		exit(0);
	}
}

/**
 * @brief Allocates host page locked memory.
 *
 * @param data Pointer to data.
 * @param mem_size How many bytes of memory to allocate
 * @return Pointer to allocated memory.
 */
void cuda_h_allocate_mem(void **data, uint64_t mem_size)
{
	//MAPPED memory test - FAILURE!
/*	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	hipSetDeviceFlags(hipDeviceMapHost);

	if (!prop.canMapHostMemory) {
		printf("[allocate_mem]: Cannot allocate host-device mapped memory. Exitting!\n");
		exit(0);
	}*/
	hipHostAlloc(data, mem_size, hipHostMallocMapped);

//	println_var(INFO, "allocating: %i [kB]\n", mem_size/1024);
	/*hipHostAlloc(data, mem_size, hipHostMallocPortable);*/

	checkCUDAError("cuda_h_allocate_mem");
}

/**
 * @brief Allocates memory on the device.
 *
 * @param data Pointer to data.
 * @param mem_size How many bytes of memory to allocate
 * @return Pointer to allocated memory.
 */
void cuda_d_allocate_mem(void **data, uint64_t mem_size)
{
//	println_var(INFO, "mem_size: %d", mem_size);
	hipMalloc(data, mem_size);
	checkCUDAError("cuda_d_allocate_mem");
}

void cuda_memcpy_hth(void *src, void *dst, uint64_t size) {
	hipMemcpy(dst, src, size, hipMemcpyHostToHost);
	checkCUDAError("cuda_memcpy_hth");
}

void cuda_memcpy_htd(void *src, void *dst, uint64_t size) {
	hipMemcpyAsync(dst, src, size, hipMemcpyHostToDevice);
	checkCUDAError("cuda_memcpy_htd");
}

void cuda_memcpy_dtd(void *src, void *dst, uint64_t size) {
	hipMemcpy(dst, src, size, hipMemcpyDeviceToDevice);
	checkCUDAError("cuda_memcpy_dtd");
}

void cuda_memcpy_dth(void *src, void *dst, uint64_t size) {
	hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
	checkCUDAError("cuda_memcpy_dth");
}

void cuda_memcpy2d_dtd(void *src, size_t src_width, void *dst, size_t dst_width, size_t width, size_t height) {
	hipMemcpy2D(dst, dst_width, src, src_width, width, height, hipMemcpyDeviceToDevice);
	checkCUDAError("cuda_memcpy2d_dtd");
}

void cuda_memcpy2d_dth(void *src, size_t src_width, void *dst, size_t dst_width, size_t width, size_t height) {
	hipMemcpy2D(dst, dst_width, src, src_width, width, height, hipMemcpyDeviceToHost);
	checkCUDAError("cuda_memcpy2d_dtd");
}

void cuda_memcpy2d_htd(void *src, size_t src_width, void *dst, size_t dst_width, size_t width, size_t height) {
	hipMemcpy2D(dst, dst_width, src, src_width, width, height, hipMemcpyHostToDevice);
	checkCUDAError("cuda_memcpy2d_dtd");
}

void cuda_d_memset(void *dst, int val, uint64_t mem_size) {
	hipMemset(dst, val, mem_size);
	checkCUDAError("cuda_d_memset");
}

void cuda_set_device(int i) {
	hipSetDevice(i);
	checkCUDAError("cuda_set_device");
}

void cuda_set_printf_limit(size_t memSize) {
	hipDeviceSetLimit(hipLimitPrintfFifoSize, memSize);
	checkCUDAError("cuda_set_printf_limit");
}
