/**
 * @file memory_management.cu
 *
 * @brief CUDA memory management functions wrappers.
 *
 * This a collection of wrappers for CUDA MM that include debugging information logging and error checking.
 *
 * @author Miłosz Ciżnicki
 * @author Jakub Misiorny <misiorny@man.poznan.pl>
 */
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdint.h>
#include <stdio.h>


#include "cuda_errors.h"
extern "C" {
	#include "memory_management.cuh"
#include "../print_info/print_info.h"
}

void cuda_d_free(void *data)
{
	hipFree(data);
	checkCUDAError("cuda_d_free");
}

void cuda_h_free(void *data) {
	hipHostFree(data);
//	checkCUDAError("cuda_h_free");
}

void cuda_set_device_flags() {
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	hipSetDeviceFlags(hipDeviceMapHost);

	if (!prop.canMapHostMemory) {
		printf("[allocate_mem]: Cannot allocate host-device mapped memory. Exitting!\n");
		exit(0);
	}
}

/**
 * @brief Allocates host page locked memory.
 *
 * @param data Pointer to data.
 * @param mem_size How many bytes of memory to allocate
 * @return Pointer to allocated memory.
 */
void cuda_h_allocate_mem(void **data, uint64_t mem_size)
{
	//MAPPED memory test - FAILURE!
/*	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	hipSetDeviceFlags(hipDeviceMapHost);

	if (!prop.canMapHostMemory) {
		printf("[allocate_mem]: Cannot allocate host-device mapped memory. Exitting!\n");
		exit(0);
	}*/
	hipHostAlloc(data, mem_size, hipHostMallocMapped);

//	println_var(INFO, "allocating: %i [kB]\n", mem_size/1024);
	/*hipHostAlloc(data, mem_size, hipHostMallocPortable);*/

	checkCUDAError("cuda_h_allocate_mem");
}

/**
 * @brief Allocates memory on the device.
 *
 * @param data Pointer to data.
 * @param mem_size How many bytes of memory to allocate
 * @return Pointer to allocated memory.
 */
void cuda_d_allocate_mem(void **data, uint64_t mem_size)
{
//	println_var(INFO, "mem_size: %d", mem_size);
	hipMalloc(data, mem_size);
	checkCUDAError("cuda_d_allocate_mem");
}

void cuda_memcpy_hth(void *src, void *dst, uint64_t size) {
	hipMemcpy(dst, src, size, hipMemcpyHostToHost);
	checkCUDAError("cuda_memcpy_hth");
}

void cuda_memcpy_htd(void *src, void *dst, uint64_t size) {
	hipMemcpyAsync(dst, src, size, hipMemcpyHostToDevice);
	checkCUDAError("cuda_memcpy_htd");
}

void cuda_memcpy_dtd(void *src, void *dst, uint64_t size) {
	hipMemcpy(dst, src, size, hipMemcpyDeviceToDevice);
	checkCUDAError("cuda_memcpy_dtd");
}

void cuda_memcpy_dth(void *src, void *dst, uint64_t size) {
	hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
	checkCUDAError("cuda_memcpy_dth");
}

void cuda_memcpy2d_dtd(void *src, size_t src_width, void *dst, size_t dst_width, size_t width, size_t height) {
	hipMemcpy2D(dst, dst_width, src, src_width, width, height, hipMemcpyDeviceToDevice);
	checkCUDAError("cuda_memcpy2d_dtd");
}

void cuda_memcpy2d_dth(void *src, size_t src_width, void *dst, size_t dst_width, size_t width, size_t height) {
	hipMemcpy2D(dst, dst_width, src, src_width, width, height, hipMemcpyDeviceToHost);
	checkCUDAError("cuda_memcpy2d_dtd");
}

void cuda_memcpy2d_htd(void *src, size_t src_width, void *dst, size_t dst_width, size_t width, size_t height) {
	hipMemcpy2D(dst, dst_width, src, src_width, width, height, hipMemcpyHostToDevice);
	checkCUDAError("cuda_memcpy2d_dtd");
}

void cuda_d_memset(void *dst, int val, uint64_t mem_size) {
	hipMemset(dst, val, mem_size);
	checkCUDAError("cuda_d_memset");
}

void cuda_set_device(int i) {
	hipSetDevice(i);
	checkCUDAError("cuda_set_device");
}

void cuda_set_printf_limit(size_t memSize) {
	hipDeviceSetLimit(hipLimitPrintfFifoSize, memSize);
	checkCUDAError("cuda_set_printf_limit");
}
