/**
 * @file cuda_errors.cu
 *
 * @author Jakub Misiorny <misiorny@man.poznan.pl>
 */

#include "cuda_errors.h"
#include <stdio.h>

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}
