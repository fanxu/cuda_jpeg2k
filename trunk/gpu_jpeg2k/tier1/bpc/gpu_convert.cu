#include "hip/hip_runtime.h"
/*
 * gpu_convert.cu
 *
 *  Created on: Feb 20, 2012
 *      Author: miloszc
 */

#include <stdio.h>
#include <assert.h>
#include "gpu_bpc.h"

#define POS_NEW_BITPOS 31
#define POS_NEW (1 << POS_NEW_BITPOS)
#define POS 0x1fff
#define EMPTY_BITPOS 30
#define EMPTY (1 << EMPTY_BITPOS)

//pos
//31		12-0
//pos_new	position

/*template <char Code_Block_Size_X>
__device__ void preprocessing(unsigned int cxds[][Code_Block_Size_X], unsigned int pos[][Code_Block_Size_X], int *blockVote, unsigned char &pass) {
//	int *blockVote_ = blockVote;
	if(TID == 0) atomicAnd(blockVote, 0);
	//blockVote = 0;
	__syncthreads();
	assert(*blockVote == 0);
	__syncthreads();
	pos[TIDY][TIDX] = TIDY * Code_Block_Size_X + TIDX;
	__syncthreads();
	// Set nbh
	unsigned int nbh = (!((cxds[TIDY][TIDX] & pass) && 1)) << POS_NEW_BITPOS;
	__syncthreads();
	// set pos_new and substrac 1 from pos if nbh > 0
	pos[TIDY][TIDX] = ((pos[TIDY][TIDX] & POS) - (nbh >> POS_NEW_BITPOS)) | nbh;
	__syncthreads();

	int warpVote = __any(nbh);
	__syncthreads();
	// voting across the blocks
	if((TID & (32 - 1)) == 0) atomicOr(blockVote, warpVote);
	__syncthreads();

	while(*blockVote) {
		// first thread of a block will reset the blockVote
		if(TID == 0) atomicAnd(blockVote, 0);
		__syncthreads();
		assert(*blockVote == 0);
		__syncthreads();
		warpVote = 0; // reset warpVote to zero
		// Get the predecessing neighbour pos state variable
		nbh = ((*(pos + TIDY * Code_Block_Size_X + TIDX - 1)) & POS_NEW);
		__syncthreads();

		pos[TIDY][TIDX] = pos[TIDY][TIDX] & (~POS_NEW);
		__syncthreads();
		// substrac 1 if nbh > 0 and jest pos_new
		pos[TIDY][TIDX] = (((pos[TIDY][TIDX] & POS) - (nbh >> POS_NEW_BITPOS)) & POS) | nbh;
		__syncthreads();
		// Voting
		warpVote = __any(nbh);
		__syncthreads();
		// execute it for the first thread of every warp only
		if((TID & (32 - 1)) == 0) atomicOr(blockVote, warpVote);
		__syncthreads();
	}
}*/

/*template <char Code_Block_Size_X>
__global__ void g_convert(CodeBlockAdditionalInfo *infos, unsigned int *g_icxds, unsigned char *g_ocxds, const int maxOutLength) {
	__shared__ unsigned int cxds[Code_Block_Size_X * Code_Block_Size_X];
//	__shared__ unsigned int pos[Code_Block_Size_X][Code_Block_Size_X];
	CodeBlockAdditionalInfo *info = &(infos[blockIdx.x]);
	int size = Code_Block_Size_X * Code_Block_Size_X;

//	if((TIDX >= info->width) || (TIDY >= info->height)) return;

	int curr_pos = 0;
	cxds[TIDY * Code_Block_Size_X + TIDX] = g_icxds[blockIdx.x * maxOutLength + TIDY * Code_Block_Size_X + TIDX];
	__syncthreads();
	if(TID == 0) {
		for(unsigned int h = 0; h < Code_Block_Size_X; ++h) {
			for(unsigned int w = 0; w < Code_Block_Size_X; ++w) {
				if(cxds[h * Code_Block_Size_X + w] & CUP) {
					unsigned char counter = cxds[h * Code_Block_Size_X + w] & CXD_COUNTER;
					for(unsigned char k = 0; k < counter; ++k) {
						unsigned char d = (cxds[h * Code_Block_Size_X + w] >> (D1_BITPOS - k * 6)) & 0x1;
						unsigned char cx = (cxds[h * Code_Block_Size_X + w] >> (CX1_BITPOS - k * 6)) & 0x1f;
//						printf("%d) %d %d\n", curr_pos, d, cx);
						g_ocxds[blockIdx.x * maxOutLength * 4 + curr_pos] = (d << 5) | cx;
						++curr_pos;
					}
				}
			}
		}
	}
	__syncthreads();

	for(unsigned char i = 1; i < info->significantBits; ++i) {
		cxds[TIDY * Code_Block_Size_X + TIDX] = g_icxds[blockIdx.x * maxOutLength + i * size + TIDY * Code_Block_Size_X + TIDX];
		__syncthreads();
		if(TID == 0) {
			for(unsigned char pass = SPP; pass < (CUP << 1); pass <<= 1) {
				for(unsigned int h = 0; h < Code_Block_Size_X; ++h) {
					for(unsigned int w = 0; w < Code_Block_Size_X; ++w) {
						if(cxds[h * Code_Block_Size_X + w] & pass) {
							unsigned char counter = cxds[h * Code_Block_Size_X + w] & CXD_COUNTER;
							for(unsigned char k = 0; k < counter; ++k) {
								unsigned char d = (cxds[h * Code_Block_Size_X + w] >> (D1_BITPOS - k * 6)) & 0x1;
								unsigned char cx = (cxds[h * Code_Block_Size_X + w] >> (CX1_BITPOS - k * 6)) & 0x1f;
//								printf("%d) %d %d\n", curr_pos, d, cx);
								g_ocxds[blockIdx.x * maxOutLength * 4 + curr_pos] = (d << 5) | cx;
								++curr_pos;
							}
						}
					}
				}
			}
		}
		__syncthreads();
	}
	if(TID == 0) {
		info->magconOffset = curr_pos;
	}
}*/

/*template <char Code_Block_Size_X, char Pass, char Bitpos>
__device__ void pass(CodeBlockAdditionalInfo *infos, unsigned int cxds[][Code_Block_Size_X + BORDER], unsigned char ocxds[(Code_Block_Size_X * 4) * (Code_Block_Size_X * 4)], unsigned int pos[][Code_Block_Size_X],
		unsigned int par_sum[Code_Block_Size_X], unsigned int sum[Code_Block_Size_X], unsigned int &offset, unsigned char *g_ocxds, const int maxOutLength) {
//	ocxds[TIDY * Code_Block_Size_X * 4 + TIDX * 4 + 0] = 0;
//	ocxds[TIDY * Code_Block_Size_X * 4 + TIDX * 4 + 1] = 0;
//	ocxds[TIDY * Code_Block_Size_X * 4 + TIDX * 4 + 2] = 0;
//	ocxds[TIDY * Code_Block_Size_X * 4 + TIDX * 4 + 3] = 0;
	if(TIDY == 0) {
		par_sum[TIDX] = 0;
//#pragma unroll 16
		for(int i = BORDER; i < Code_Block_Size_X + BORDER; ++i) {
			par_sum[TIDX] += ((cxds[X][i] & Pass) >> Bitpos) * (cxds[X][i] & CXD_COUNTER);
		}
	}
	__syncthreads();

	if(TIDY == 0) {
		sum[TIDX] = offset;
//		if(offset > 30000)
//		printf("%x	%d\n", sum[TIDX], TIDX);
		for(int i = 0; i < TIDX; ++i) {
			sum[TIDX] += par_sum[i];
		}
	}
	__syncthreads();

	if(TIDY == 0) {
//		printf("%x	%d\n", sum[TIDX], TIDX);
		unsigned int curr_pos = sum[TIDX];
//		if(curr_pos > 3000)
//		printf("%x 	%d	%d\n", curr_pos, TIDX, 0);
//#pragma unroll 16
		for(int i = 0; i < Code_Block_Size_X; ++i) {
			curr_pos += ((cxds[X][i] & Pass) >> Bitpos) * (cxds[X][i] & CXD_COUNTER);
			pos[TIDX][i] = curr_pos;
//			if(pos[TIDX][i] > 3000)
//				printf("%d 	%d	%d\n", pos[TIDX][i], TIDX, i);
		}
	}
	__syncthreads();
//	if((TIDY == (Code_Block_Size_X - 1)) && (TIDX == (Code_Block_Size_X - 1))) {
//		printf("size %d pos %d\n", offset, pos[TIDY][TIDX]);
//	}
//	__syncthreads();

	for(unsigned char k = 0; k < (cxds[Y][X] & CXD_COUNTER) * ((cxds[Y][X] & Pass) >> Bitpos); ++k) {
//		ocxds[pos[TIDY][TIDX] - offset + k] =
		g_ocxds[blockIdx.x * maxOutLength * 4 + pos[TIDY][TIDX] + k] =
				(((cxds[Y][X] >> (D1_BITPOS - k * 6)) & 0x1) << 5) | ((cxds[Y][X] >> (CX1_BITPOS - k * 6)) & 0x1f);
//		printf("%x	%x	%d	%d	%d	%d	%d\n", cxds[Y][X], Pass, ((cxds[Y][X] >> (D1_BITPOS - k * 6)) & 0x1), ((cxds[Y][X] >> (CX1_BITPOS - k * 6)) & 0x1f), blockIdx.x * maxOutLength * 4 + pos[TIDY][TIDX] + k, TIDY, TIDX);
//		printf("%x	%d	%d	%d	%d	%d\n", ocxds[pos[TIDY][TIDX] + k], ((cxds[Y][X] >> (D1_BITPOS - k * 6)) & 0x1), ((cxds[Y][X] >> (CX1_BITPOS - k * 6)) & 0x1f), pos[TIDY][TIDX] + k, TIDY, TIDX);
	}
	__syncthreads();
//	printf("write %d	%d	%d	%d\n", blockIdx.x * maxOutLength * 4 + offset + TIDY * Code_Block_Size_X * 4 + TIDX * 4, offset, TIDY, TIDX);
//	g_ocxds[blockIdx.x * maxOutLength * 4 + offset + TIDY * Code_Block_Size_X * 4 + TIDX * 4 + 0] = ocxds[TIDY * Code_Block_Size_X * 4 + TIDX * 4 + 0];
//	g_ocxds[blockIdx.x * maxOutLength * 4 + offset + TIDY * Code_Block_Size_X * 4 + TIDX * 4 + 1] = ocxds[TIDY * Code_Block_Size_X * 4 + TIDX * 4 + 1];
//	g_ocxds[blockIdx.x * maxOutLength * 4 + offset + TIDY * Code_Block_Size_X * 4 + TIDX * 4 + 2] = ocxds[TIDY * Code_Block_Size_X * 4 + TIDX * 4 + 2];
//	g_ocxds[blockIdx.x * maxOutLength * 4 + offset + TIDY * Code_Block_Size_X * 4 + TIDX * 4 + 3] = ocxds[TIDY * Code_Block_Size_X * 4 + TIDX * 4 + 3];
	__syncthreads();

//	if(TID == 0) printf("offset %d	%d	%d\n", offset, TIDY, TIDX);

	if((TIDY == (Code_Block_Size_X - 1)) && (TIDX == (Code_Block_Size_X - 1))) {
		offset = pos[TIDY][TIDX] + ((cxds[Y][X] & Pass) >> Bitpos) * (cxds[Y][X] & CXD_COUNTER);
//		printf("size %d pos %d\n", offset, pos[TIDY][TIDX]);
	}
	__syncthreads();
}*/

template <char Code_Block_Size_X, char Pass, char Bitpos>
__device__ void pass(CodeBlockAdditionalInfo *infos, unsigned int cxds[][Code_Block_Size_X + BORDER], unsigned char ocxds[(Code_Block_Size_X * 4) * (Code_Block_Size_X * 4)], unsigned int pos[][Code_Block_Size_X + BORDER],
		unsigned int sum[Code_Block_Size_X], unsigned int &offset, unsigned char *g_ocxds, const int maxOutLength) {
//	ocxds[TIDY * Code_Block_Size_X * 4 + TIDX * 4 + 0] = 0;
//	ocxds[TIDY * Code_Block_Size_X * 4 + TIDX * 4 + 1] = 0;
//	ocxds[TIDY * Code_Block_Size_X * 4 + TIDX * 4 + 2] = 0;
//	ocxds[TIDY * Code_Block_Size_X * 4 + TIDX * 4 + 3] = 0;
	pos[TIDY][TIDX] = 0;
	unsigned int curr_pos = 0;
	if(TIDY == 0) {
		for(int i = BORDER; i < Code_Block_Size_X + BORDER; ++i) {
			curr_pos += ((cxds[X][i] & Pass) >> Bitpos) * (cxds[X][i] & CXD_COUNTER);
			pos[TIDX][i] = curr_pos;
//			if(pos[TIDX][i] > 3000)
//				printf("%d 	%d	%d\n", pos[TIDX][i], TIDX, i);
		}
	}
	__syncthreads();

	if(TIDY == 0) {
		sum[TIDX] = offset;
//		if(offset > 30000)
//		printf("%x	%d\n", sum[TIDX], TIDX);
		for(int i = 0; i < TIDX; ++i) {
			sum[TIDX] += /*par_sum[i]*/pos[i][Code_Block_Size_X];
		}
	}
	__syncthreads();

	pos[TIDY][TIDX] += sum[TIDY];
	__syncthreads();

	unsigned char counter = (cxds[Y][X] & CXD_COUNTER) * ((cxds[Y][X] & Pass) >> Bitpos);

	for(unsigned char k = 0; k < counter; ++k) {
//		ocxds[pos[TIDY][TIDX] - offset + k] =
		g_ocxds[blockIdx.x * maxOutLength * 4 + pos[TIDY][TIDX] + k] = ((cxds[Y][X] >> (D1_BITPOS - k * 6)) & 0x3f);
//				(((cxds[Y][X] >> (D1_BITPOS - k * 6)) & 0x1) << 5) | ((cxds[Y][X] >> (CX1_BITPOS - k * 6)) & 0x1f);
//		printf("%x	%x	%d	%d	%d	%d	%d\n", cxds[Y][X], Pass, ((cxds[Y][X] >> (D1_BITPOS - k * 6)) & 0x1), ((cxds[Y][X] >> (CX1_BITPOS - k * 6)) & 0x1f), blockIdx.x * maxOutLength * 4 + pos[TIDY][TIDX] + k, TIDY, TIDX);
//		printf("%x	%d	%d	%d	%d	%d\n", ocxds[pos[TIDY][TIDX] + k], ((cxds[Y][X] >> (D1_BITPOS - k * 6)) & 0x1), ((cxds[Y][X] >> (CX1_BITPOS - k * 6)) & 0x1f), pos[TIDY][TIDX] + k, TIDY, TIDX);
	}
	__syncthreads();
//	printf("write %d	%d	%d	%d\n", blockIdx.x * maxOutLength * 4 + offset + TIDY * Code_Block_Size_X * 4 + TIDX * 4, offset, TIDY, TIDX);
//	g_ocxds[blockIdx.x * maxOutLength * 4 + offset + TIDY * Code_Block_Size_X * 4 + TIDX * 4 + 0] = ocxds[TIDY * Code_Block_Size_X * 4 + TIDX * 4 + 0];
//	g_ocxds[blockIdx.x * maxOutLength * 4 + offset + TIDY * Code_Block_Size_X * 4 + TIDX * 4 + 1] = ocxds[TIDY * Code_Block_Size_X * 4 + TIDX * 4 + 1];
//	g_ocxds[blockIdx.x * maxOutLength * 4 + offset + TIDY * Code_Block_Size_X * 4 + TIDX * 4 + 2] = ocxds[TIDY * Code_Block_Size_X * 4 + TIDX * 4 + 2];
//	g_ocxds[blockIdx.x * maxOutLength * 4 + offset + TIDY * Code_Block_Size_X * 4 + TIDX * 4 + 3] = ocxds[TIDY * Code_Block_Size_X * 4 + TIDX * 4 + 3];
//	__syncthreads();

//	if(TID == 0) printf("offset %d	%d	%d\n", offset, TIDY, TIDX);

	if((TIDY == (Code_Block_Size_X - 1)) && (TIDX == (Code_Block_Size_X - 1))) {
		offset = pos[TIDY][TIDX] + counter;
//		printf("size %d pos %d\n", offset, pos[TIDY][TIDX]);
	}
	__syncthreads();
}

template <char Code_Block_Size_X>
__global__ void g_convert(CodeBlockAdditionalInfo *infos, unsigned int *g_icxds, unsigned char *g_ocxds, const int maxOutLength) {
	__shared__ unsigned int cxds[Code_Block_Size_X + BORDER][Code_Block_Size_X + BORDER];
	__shared__ unsigned char ocxds[(Code_Block_Size_X * Code_Block_Size_X * 16)];
	__shared__ unsigned int pos[Code_Block_Size_X][Code_Block_Size_X + BORDER];
	__shared__ unsigned int sum[Code_Block_Size_X];
	__shared__ unsigned int offset;
	CodeBlockAdditionalInfo *info = &(infos[blockIdx.x]);
	int size = Code_Block_Size_X * Code_Block_Size_X;

//	if(info->significantBits == 0) return;

	cxds[Y][X] = g_icxds[blockIdx.x * maxOutLength + TIDY * Code_Block_Size_X + TIDX];
//	printf("%x	%d	%d\n", ((cxds[Y][X] & SPP) >> SPP_BITPOS) * (cxds[Y][X] & CXD_COUNTER), TIDY, TIDX);
	if(TID == 0) offset = 0;
	if(TIDY == 0) cxds[X][0] = 0;
	__syncthreads();

	pass<Code_Block_Size_X, CUP, CUP_BITPOS>(info, cxds, ocxds, pos, sum, offset, g_ocxds, maxOutLength);
	__syncthreads();

	for(unsigned char i = 1; i < info->significantBits; ++i) {
		cxds[Y][X] = g_icxds[blockIdx.x * maxOutLength + i * size + TIDY * Code_Block_Size_X + TIDX];
//		if(TIDY == 0) cxds[TIDX][0] = 0;
		__syncthreads();
//		if(TID == 0) printf("%d) offset %d	%d	%d\n", i, offset, TIDY, TIDX);
		pass<Code_Block_Size_X, SPP, SPP_BITPOS>(info, cxds, ocxds, pos, sum, offset, g_ocxds, maxOutLength);
		__syncthreads();
//		if(TID == 0) printf("%d) offset %d	%d	%d\n", i, offset, TIDY, TIDX);
		pass<Code_Block_Size_X, MRP, MRP_BITPOS>(info, cxds, ocxds, pos, sum, offset, g_ocxds, maxOutLength);
		__syncthreads();
//		if(TID == 0) printf("%d) offset %d	%d	%d\n", i, offset, TIDY, TIDX);
		pass<Code_Block_Size_X, CUP, CUP_BITPOS>(info, cxds, ocxds, pos, sum, offset, g_ocxds, maxOutLength);
		__syncthreads();
	}

	if(TID == 0) info->magconOffset = offset;
}

void convert(dim3 gridDim, dim3 blockDim, CodeBlockAdditionalInfo *infos, unsigned int *g_icxds, unsigned char *g_ocxds, const int maxOutLength)
{
	printf("dim %d %d\n", blockDim.x, blockDim.y);
//	hipFuncSetCacheConfig(reinterpret_cast<const void*>(g_convert<16>), hipFuncCachePreferL1);
	switch(blockDim.x) {
//	case 4: g_convert<4><<<gridDim, blockDim>>>(infos, g_icxds, g_ocxds, maxOutLength); break;
//	case 8: g_convert<8><<<gridDim, blockDim>>>(infos, g_icxds, g_ocxds, maxOutLength); break;
	case 16: g_convert<16><<<gridDim, blockDim>>>(infos, g_icxds, g_ocxds, maxOutLength); break;
	case 32: g_convert<32><<<gridDim, blockDim>>>(infos, g_icxds, g_ocxds, maxOutLength); break;
//	case 64: bpc_encoder<64><<<gridDim, blockDim>>>(infos, g_cxds); break;
	}

	hipDeviceSynchronize();
	hipError_t cuerr;
	if (cuerr = hipGetLastError()) {
		printf("bpc_encoder error: %s\n", hipGetErrorString(cuerr));
		return;
	}
}
