#include "hip/hip_runtime.h"
/*
 * @file vector_cuda.cu
 *
 * @author Milosz Ciznicki 
 * @date 06-05-2011
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "vector.h"

static __global__ void vector_mult_cuda(float *val, unsigned n,
		float factor)
{
	unsigned i;
	for(i = 0; i < n; i++)
		val[i] *= factor;
}

extern "C" void scal_cuda_func(void *data_interface)
{
	data *data_i = (data *)data_interface;
	vector *vec = data_i->vec;
	float factor = data_i->factor;

	/* length of the vector */
	unsigned n = vec->size;

	float *d_array;

	hipMalloc((void **)&d_array, n * sizeof(float));

	hipMemcpy(d_array, vec->array, n * sizeof(float), hipMemcpyHostToDevice);

	/* TODO: use more blocks and threads in blocks */
	vector_mult_cuda<<<1,1>>>(d_array, n, factor);

	hipMemcpy(vec->array, d_array, n * sizeof(float), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
}
