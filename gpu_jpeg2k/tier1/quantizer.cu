#include "hip/hip_runtime.h"
/**
 * @file quantization.cu
 *
 * @author Milosz Ciznicki
 * @date 06-09-2010
 */

#include "../types/image_types.h"
#include "../misc/cuda_errors.h"
extern "C" {
	#include "quantizer.h"
	#include "quantization.h"
	#include "../misc/memory_management.cuh"
	#include "../print_info/print_info.h"
}

/**
 * @brief Subband quantization.
 *
 * @param idata Input tile_comp_data.
 * @param size Width and height of subbnad.
 * @param step_size Step size(deltab).
 */
__global__
void subband_quantization_lossy(type_data *idata, int2 isize, int *odata, int2 osize, int2 cblk_size, const float convert_factor)
{
	int cblk_sum_size = cblk_size.x * cblk_size.y;
	int block_id = blockIdx.x + blockIdx.y * gridDim.x;
	int i = threadIdx.x;
	int j = threadIdx.y;
	int n = i + blockIdx.x * cblk_size.x;
	int m = j + blockIdx.y * cblk_size.y;
	int in = n + m * isize.x;
	int out = i + j * cblk_size.x + (block_id) * cblk_sum_size;
	int cache_value;

	while(j < cblk_size.y && m < osize.y)
	{
		while(i < cblk_size.x &&  n < osize.x)
		{
			cache_value = idata[in] * convert_factor;
			odata[out] = cache_value < 0 ? (1 << 31) | ((int)(-cache_value)) : cache_value;
//			odata[out] = (idata[in] * convert_factor) < 0 ? (1 << 31) | ((int)(-(idata[in] * convert_factor))) : (idata[in] * convert_factor);
			i += BLOCKSIZEX;
			n = i + blockIdx.x * cblk_size.x;
			in = n + m * isize.x;
			out = i + j * cblk_size.x + (block_id) * cblk_sum_size;
		}
		i = threadIdx.x;
		j += BLOCKSIZEY;
		n = i + blockIdx.x * cblk_size.x;
		m = j + blockIdx.y * cblk_size.y;
		in = n + m * isize.x;
		out = i + j * cblk_size.x + (block_id) * cblk_sum_size;
	}
}

/**
 * @brief Subband quantization.
 *
 * @param idata Input tile_comp_data.
 * @param size Width and height of subbnad.
 * @param step_size Step size(deltab).
 */
__global__
void subband_quantization_lossless(type_data *idata, int2 isize, int *odata, int2 osize, int2 cblk_size, const int shift_bits)
{
	int cblk_sum_size = cblk_size.x * cblk_size.y;
	int block_id = blockIdx.x + blockIdx.y * gridDim.x;
	int i = threadIdx.x;
	int j = threadIdx.y;
	int n = i + blockIdx.x * cblk_size.x;
	int m = j + blockIdx.y * cblk_size.y;
	int in = n + m * isize.x;
	int out = i + j * cblk_size.x + (block_id) * cblk_sum_size;
	int cache_value;

	while(j < cblk_size.y && m < osize.y)
	{
		while(i < cblk_size.x &&  n < osize.x)
		{
			cache_value = ((int)idata[in]) << shift_bits;
			odata[out] = cache_value < 0 ? (1 << 31) | (-cache_value) : cache_value;
//			odata[out] = (idata[in] * convert_factor) < 0 ? (1 << 31) | ((int)(-(idata[in] * convert_factor))) : (idata[in] * convert_factor);
			i += BLOCKSIZEX;
			n = i + blockIdx.x * cblk_size.x;
			in = n + m * isize.x;
			out = i + j * cblk_size.x + (block_id) * cblk_sum_size;
		}
		i = threadIdx.x;
		j += BLOCKSIZEY;
		n = i + blockIdx.x * cblk_size.x;
		m = j + blockIdx.y * cblk_size.y;
		in = n + m * isize.x;
		out = i + j * cblk_size.x + (block_id) * cblk_sum_size;
	}
}

/**
 * @brief Subband quantization.
 *
 * @param subband Input subband.
 * @param size Width and height of subbnad.
 */
type_subband *quantization(type_subband *sb)
{
//	println_start(INFO);

	dim3 blocks;
	dim3 threads;
	type_codeblock *cblk;

	type_res_lvl *res_lvl = sb->parent_res_lvl;
	type_tile_comp *tile_comp = res_lvl->parent_tile_comp;
	type_image *img = tile_comp->parent_tile->parent_img;

	/* Base step size */
	float base_step = /*BASE_STEP*/(float)1 / (float)(1 << (img->num_range_bits - 1));
	int guard_bits = GUARD_BITS;
	/* Relative quantization step size. Step size is signaled relative to the wavelet coefficient bit depth. */
	float relative_step_size;
	int shift_bits;

	/* Lossy */
	if(img->wavelet_type)
	{
		/* The number of magnitude bits in the integer representation of the quantized data */
		sb->mag_bits = guard_bits - 1 + res_lvl->dec_lvl_no - (int)(log(base_step) / log(2));
		/* Relative quantization step size. Step size is signaled relative to the wavelet coefficient bit depth. */
		relative_step_size = base_step / (1 << res_lvl->dec_lvl_no);
		shift_bits = 31 - sb->mag_bits;

		/* Calculate step size and convert factor for quantization */
		relative_step_size = convert_from_exp_mantissa(convert_to_exp_mantissa(relative_step_size));
		sb->step_size = ((1L << (img->num_range_bits + get_exp_subband_gain(sb->orient))) * relative_step_size);
		sb->convert_factor = (1.0f / sb->step_size) * (1 << (shift_bits));

//		println_var(INFO, "Lossy mag_bits:%d convert_factor:%f shift_bits:%d relative_step_size:%f step_size:%f subband_gain:%d", sb->mag_bits, sb->convert_factor, shift_bits, relative_step_size, sb->step_size, get_exp_subband_gain(sb->orient));

	} else /* Lossless */
	{
		/* K_b_max = B - 1 + X_b + G */
		sb->mag_bits = img->num_range_bits - 1 + get_exp_subband_gain(sb->orient) + guard_bits;
		shift_bits = 31 - sb->mag_bits;
		//printf("magbits %d\n", sb->mag_bits);
		sb->convert_factor = 1 << shift_bits;

//		println_var(INFO, "Lossless mag_bits:%d convert_factor:%f shift_bits:%d subband_gain:%d", sb->mag_bits, sb->convert_factor, shift_bits, get_exp_subband_gain(sb->orient));
	}
//	println_var(INFO, "%d %d %d %d %d %d", sb->width, sb->num_cblks, sb->num_xcblks, sb->num_ycblks, tile_comp->cblk_w, tile_comp->cblk_h);
	mem_mg_t *mem_mg = img->mem_mg;
	/* Allocate aligned memory for linear codeblocks data */
	sb->cblks_data_d = (int32_t *)mem_mg->alloc->dev(sb->num_cblks * tile_comp->cblk_w * tile_comp->cblk_h * sizeof(int32_t), mem_mg->ctx);

	/* Input and output data */
	type_data *idata = tile_comp->img_data_d + sb->tlx + sb->tly * tile_comp->width;
	int *odata = sb->cblks_data_d;
	int2 isize = make_int2(tile_comp->width, tile_comp->height);
	int2 osize = make_int2(sb->width, sb->height);
	int2 cblk_size = make_int2(tile_comp->cblk_w, tile_comp->cblk_h);

	/* Number of blocks for parallel reduction */
	blocks = dim3(sb->num_xcblks , sb->num_ycblks);
	/* Number of threads for parallel reduction */
	threads = dim3(BLOCKSIZEX, BLOCKSIZEY);

//	println_var(INFO, "blocks.x:%d blocks.y:%d threads.x:%d threads.y:%d", blocks.x, blocks.y, threads.x, threads.y);
//	println_var(INFO, "isize.x:%d isize.y:%d osize.x:%d osize.y:%d cblk_size.x:%d cblk_size.y:%d", isize.x, isize.y, osize.x, osize.y, cblk_size.x, cblk_size.y);
	if(img->wavelet_type)
	{
		subband_quantization_lossy<<<blocks, threads>>>(idata, isize, odata, osize, cblk_size, sb->convert_factor);
	} else
	{
		subband_quantization_lossless<<<blocks, threads>>>(idata, isize, odata, osize, cblk_size, shift_bits);
	}

	hipDeviceSynchronize();

	checkCUDAError("quantization");

	unsigned int i;
	/* Update code blocks pointer to data */
	for(i = 0; i < sb->num_cblks; i++)
	{
		cblk = &(sb->cblks[i]);
		cblk->data_d = sb->cblks_data_d + cblk->cblk_no * tile_comp->cblk_w * tile_comp->cblk_h;
	}

//	println_end(INFO);

	return sb;
}

/**
 * @brief Do quantization for every subbands from tile.
 * @param tile
 */
void quantize_tile(type_tile *tile)
{
//	println_start(INFO);

//	start_measure();

	type_image *img = tile->parent_img;
	mem_mg_t *mem_mg = img->mem_mg;
	type_tile_comp *tile_comp;
	type_res_lvl *res_lvl;
	type_subband *sb;
	int i, j, k;

	for(i = 0; i < img->num_components; i++)
	{
		tile_comp = &(tile->tile_comp[i]);
		for(j = 0; j < tile_comp->num_rlvls; j++)
		{
			res_lvl = &(tile_comp->res_lvls[j]);
			for(k = 0; k < res_lvl->num_subbands; k++)
			{
				sb = &(res_lvl->subbands[k]);
				quantization(sb);
			}
		}
		mem_mg->dealloc->dev(tile_comp->img_data_d, mem_mg->ctx);
	}

//	stop_measure(INFO);

//	println_end(INFO);
}

/**
 * @brief Do quantization for every subbands from tile.
 * @param tile
 */
/*void quantize_tile_dbg(type_tile *tile)
{
	println_start(INFO);

	start_measure();

	type_image *img = tile->parent_img;
	type_tile_comp *tile_comp;
	type_res_lvl *res_lvl;
	type_subband *sb;
	int i, j, k;

	int x = 0, y = 0;
	int size = tile->width * tile->height * sizeof(type_data);
	type_data *buff = (type_data*)malloc(size);

	for(i = 0; i < img->num_components; i++)
	{
		tile_comp = &(tile->tile_comp[i]);
		cuda_memcpy_dth(tile_comp->img_data_d, buff, size);

		for(y = 0; y < tile_comp->height; y++)
		{
			for(x = 0; x < tile_comp->width; x++)
			{
				printf("%f, ", buff[x + y * tile_comp->width]);
			}
		}
		printf("\n");

		for(j = 0; j < tile_comp->num_rlvls; j++)
		{
			res_lvl = &(tile_comp->res_lvls[j]);
			for(k = 0; k < res_lvl->num_subbands; k++)
			{
				sb = &(res_lvl->subbands[k]);
				quantization(sb);
			}
		}
		cuda_d_free(tile_comp->img_data_d);
	}

	stop_measure(INFO);

	println_end(INFO);
}*/

/**
 * @brief Subband quantization.
 *
 * @param subband Input subband.
 * @param size Width and height of subbnad.
 */
//type_subband *quantization_dbg(type_subband *sb)
//{
////	println_start(INFO);
//
//	dim3 blocks;
//	dim3 threads;
//	int i;
//	type_codeblock *cblk;
//
//	type_res_lvl *res_lvl = sb->parent_res_lvl;
//	type_tile_comp *tile_comp = res_lvl->parent_tile_comp;
//	type_image *img = tile_comp->parent_tile->parent_img;
//
//	/* Base step size */
//	float base_step = BASE_STEP;
//	int guard_bits = GUARD_BITS;
//	/* Relative quantization step size. Step size is signaled relative to the wavelet coefficient bit depth. */
//	float relative_step_size;
//	int shift_bits;
//
//	/* Lossy */
//	if(img->wavelet_type)
//	{
//		/* The number of magnitude bits in the integer representation of the quantized data */
//		sb->mag_bits = guard_bits - 1 + res_lvl->dec_lvl_no - (int)(log(base_step) / log(2));
//		/* Relative quantization step size. Step size is signaled relative to the wavelet coefficient bit depth. */
//		relative_step_size = base_step / (1 << res_lvl->dec_lvl_no);
//		shift_bits = 31 - sb->mag_bits;
//
//		/* Calculate step size and convert factor for quantization */
//		relative_step_size = convert_from_exp_mantissa(convert_to_exp_mantissa(relative_step_size));
//		sb->step_size = ((1L << (img->nom_range_bits + get_exp_subband_gain(sb->orient))) * relative_step_size);
//		sb->convert_factor = (1.0f / sb->step_size) * (1 << (shift_bits));
//
////		println_var(INFO, "Lossy mag_bits:%d convert_factor:%f shift_bits:%d relative_step_size:%f step_size:%f subband_gain:%d", sb->mag_bits, sb->convert_factor, shift_bits, relative_step_size, sb->step_size, get_exp_subband_gain(sb->orient));
//
//	} else /* Lossless */
//	{
//		/* K_b_max = B - 1 + X_b + G */
//		sb->mag_bits = img->nom_range_bits - 1 + get_exp_subband_gain(sb->orient) + guard_bits;
//		shift_bits = 31 - sb->mag_bits;
//		sb->convert_factor = 1 << shift_bits;
//
////		println_var(INFO, "Lossless mag_bits:%d convert_factor:%f shift_bits:%d subband_gain:%d", sb->mag_bits, sb->convert_factor, shift_bits, get_exp_subband_gain(sb->orient));
//	}
////	println_var(INFO, "%d %d %d %d %d %d", sb->width, sb->num_cblks, tile_comp->cblk_w, tile_comp->cblk_h, sizeof(long int), sizeof(size_t));
//	/* Allocate aligned memory for linear codeblocks data */
//	cuda_d_allocate_mem((void **) &(sb->cblks_data_d), sb->num_cblks * tile_comp->cblk_w * tile_comp->cblk_h * sizeof(int));
//
//	/* Input and output data */
//	type_data *idata = tile_comp->img_data_d + sb->tlx + sb->tly * tile_comp->height;
//	int *odata = sb->cblks_data_d;
//	int2 isize = make_int2(tile_comp->width, tile_comp->height);
//	int2 osize = make_int2(sb->width, sb->height);
//	int2 cblk_size = make_int2(tile_comp->cblk_w, tile_comp->cblk_h);
//
//	/* Number of blocks for parallel reduction */
//	blocks = dim3(sb->num_xcblks , sb->num_ycblks);
//	/* Number of threads for parallel reduction */
//	threads = dim3(BLOCKSIZEX, BLOCKSIZEY);
//
////	quantization_dbg(idata, osize.x * osize.y * sizeof(type_data), osize.x, osize.y);
//
////	println_var(INFO, "blocks.x:%d blocks.y:%d threads.x:%d threads.y:%d", blocks.x, blocks.y, threads.x, threads.y);
////	println_var(INFO, "isize.x:%d isize.y:%d osize.x:%d osize.y:%d cblk_size.x:%d cblk_size.y:%d", isize.x, isize.y, osize.x, osize.y, cblk_size.x, cblk_size.y);
//	if(img->wavelet_type)
//	{
//		subband_quantization_lossy<<<blocks, threads>>>(idata, isize, odata, osize, cblk_size, sb->convert_factor);
//	} else
//	{
//		subband_quantization_lossless<<<blocks, threads>>>(idata, isize, odata, osize, cblk_size, shift_bits);
//	}
//
//	hipDeviceSynchronize();
//
//	checkCUDAError("quantization");
//
//	int size = tile_comp->cblk_w * tile_comp->cblk_h * sizeof(int);
//	int *buff = (int*)malloc(size);
//	int x = 0, y = 0;
//
//	/* Update code blocks pointer to data */
//	for(i = 0; i < sb->num_cblks; i++)
//	{
//		cblk = &(sb->cblks[i]);
//		cblk->data_d = sb->cblks_data_d + cblk->cblk_no * tile_comp->cblk_w * tile_comp->cblk_h;
//
//		cuda_memcpy_dth(cblk->data_d, buff, size);
//
//		printf("ulx:%d uly:%d magbits:%d orient:%d con:%f\n", sb->tlx, sb->tly, sb->mag_bits, sb->orient, sb->convert_factor);
//
//		for(y = cblk->height - 1; y >= 0; y--)
//		{
//			for(x = cblk->width - 1; x >= 0; x--)
//			{
//				printf("%d, ", buff[x + y * tile_comp->cblk_w]);
//			}
//		}
//		printf("\n");
//	}
//
////	println_end(INFO);
//
//	return sb;
//}

/*void quantization_dbg(type_data *idata, int size, int w, int h)
{
	type_data *buff = (type_data*)malloc(size);
	int x = 0, y = 0;

	cuda_memcpy_dth(idata, buff, size);

	for(y = h - 1; y >= 0; y--)
	{
		for(x = w - 1; x >= 0; x--)
		{
			printf("%f, ", buff[x + y * w]);
		}
	}
	printf("\n");
	free(buff);
}*/
