#include "hip/hip_runtime.h"
/*
 * reduce.cu
 *
 *  Created on: Nov 30, 2011
 *      Author: miloszc
 */
#include "../types/image_types.h"
extern "C" {
#include "reduce.h"
#include "blocks.h"
#include "../misc/memory_management.cuh"
}
#include "../misc/cuda_errors.h"

#define MAX_THREADS 128

static bool isPow2(unsigned int x)
{
    return ((x&(x-1))==0);
}

/*
 * @brief Reduction kernel
 * @param g_idata	device input data
 * @param g_odata	device reduced output data
 * @param n			list size
 */
template <class T, unsigned int blockSize, bool nIsPow2>
__global__ void
reduce(T *g_idata, T *g_odata, unsigned int n)
{
	__shared__ T sdata[1024];

	// perform first level of reduction,
	// reading from global memory, writing to shared memory
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
	unsigned int gridSize = blockSize*2*gridDim.x;

	T mySum = 0;

	// we reduce multiple elements per thread.  The number is determined by the
	// number of active thread blocks (via gridDim).  More blocks will result
	// in a larger gridSize and therefore fewer elements per thread
	while (i < n)
	{
		mySum += g_idata[i];
		// ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
		if (nIsPow2 || i + blockSize < n)
			mySum += g_idata[i+blockSize];
		i += gridSize;
	}

	// each thread puts its local sum into shared memory
	sdata[tid] = mySum;
	__syncthreads();

	// do reduction in shared mem
	if (blockSize >= 512) {if (tid < 256) {sdata[tid] = mySum = mySum + sdata[tid + 256];}__syncthreads();}
	if (blockSize >= 256) {if (tid < 128) {sdata[tid] = mySum = mySum + sdata[tid + 128];}__syncthreads();}
	if (blockSize >= 128) {if (tid < 64) {sdata[tid] = mySum = mySum + sdata[tid + 64];}__syncthreads();}

	{
		// now that we are using warp-synchronous programming (below)
		// we need to declare our shared memory volatile so that the compiler
		// doesn't reorder stores to it and induce incorrect behavior.
		volatile T* smem = sdata;
		if (blockSize >= 64) {smem[tid] = mySum = mySum + smem[tid + 32];}
		if (blockSize >= 32) {smem[tid] = mySum = mySum + smem[tid + 16];}
		if (blockSize >= 16) {smem[tid] = mySum = mySum + smem[tid + 8];}
		if (blockSize >= 8) {smem[tid] = mySum = mySum + smem[tid + 4];}
		if (blockSize >= 4) {smem[tid] = mySum = mySum + smem[tid + 2];}
		if (blockSize >= 2) {smem[tid] = mySum = mySum + smem[tid + 1];}
	}

	// write result for this block to global mem
	if (tid == 0)
		g_odata[blockIdx.x] = sdata[0];
}

/*
 * @brief Execute kernel with appropriate number of threads.
 */
void graph_reduce(int threads, int blocks, type_data *d_idata, type_data *d_odata, int size) {
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);

	if (isPow2(size)) {
		switch (threads) {
		case 512:
			reduce<type_data, 512, true><<< dimGrid, dimBlock>>>(d_idata, d_odata, size); break;
		case 256:
			reduce<type_data, 256, true><<< dimGrid, dimBlock>>>(d_idata, d_odata, size); break;
		case 128:
			reduce<type_data, 128, true><<< dimGrid, dimBlock>>>(d_idata, d_odata, size); break;
		case 64:
			reduce<type_data, 64, true><<< dimGrid, dimBlock>>>(d_idata, d_odata, size); break;
		case 32:
			reduce<type_data, 32, true><<< dimGrid, dimBlock>>>(d_idata, d_odata, size); break;
		case 16:
			reduce<type_data, 16, true><<< dimGrid, dimBlock>>>(d_idata, d_odata, size); break;
		case 8:
			reduce<type_data, 8, true><<< dimGrid, dimBlock>>>(d_idata, d_odata, size); break;
		case 4:
			reduce<type_data, 4, true><<< dimGrid, dimBlock>>>(d_idata, d_odata, size); break;
		case 2:
			reduce<type_data, 2, true><<< dimGrid, dimBlock>>>(d_idata, d_odata, size); break;
		case 1:
			reduce<type_data, 1, true><<< dimGrid, dimBlock>>>(d_idata, d_odata, size); break;
		}
	} else {
		switch (threads) {
		case 512:
			reduce<type_data, 512, false><<< dimGrid, dimBlock>>>(d_idata, d_odata, size); break;
		case 256:
			reduce<type_data, 256, false><<< dimGrid, dimBlock>>>(d_idata, d_odata, size); break;
		case 128:
			reduce<type_data, 128, false><<< dimGrid, dimBlock>>>(d_idata, d_odata, size); break;
		case 64:
			reduce<type_data, 64, false><<< dimGrid, dimBlock>>>(d_idata, d_odata, size); break;
		case 32:
			reduce<type_data, 32, false><<< dimGrid, dimBlock>>>(d_idata, d_odata, size); break;
		case 16:
			reduce<type_data, 16, false><<< dimGrid, dimBlock>>>(d_idata, d_odata, size); break;
		case 8:
			reduce<type_data, 8, false><<< dimGrid, dimBlock>>>(d_idata, d_odata, size); break;
		case 4:
			reduce<type_data, 4, false><<< dimGrid, dimBlock>>>(d_idata, d_odata, size); break;
		case 2:
			reduce<type_data, 2, false><<< dimGrid, dimBlock>>>(d_idata, d_odata, size); break;
		case 1:
			reduce<type_data, 1, false><<< dimGrid, dimBlock>>>(d_idata, d_odata, size); break;
		}
	}
}

/*
 * @brief Reduce graph data.
 * @param device graph input data
 * @param width and height of graph data
 * @return device reduced data
 */
type_data reduction(type_data *d_i_data, type_data *d_o_data, type_data *h_odata, int size) {
	int maxThreads = MAX_THREADS; /* number of threads per block */
	int maxBlocks = MAX_BLOCKS; /* maximum number of blocks used for reducition */
	int numBlocks = 0; /* default number of blocks used for reducing one list */
	int numThreads = 0; /* default number of threads */
	dim3 threads, blocks;
	type_data gpu_result = 0.0f;

	/* calculate how many blocks and threads use for reduction kernel */
	get_num_blocks_and_threads(size, maxBlocks, maxThreads, &numBlocks, &numThreads);

//	printf("%d, %d, %d, %d, %d\n", size, maxBlocks, maxThreads, numBlocks, numThreads);

	/* sum up all */
	graph_reduce(numThreads, numBlocks, d_i_data, d_o_data, size);
	checkCUDAError("\tafter reduce");

	cuda_memcpy_dth(d_o_data, h_odata, numBlocks * sizeof(type_data));

	for(int i=0; i<numBlocks; i++)
	{
		gpu_result += h_odata[i];
	}

	return gpu_result;
}
