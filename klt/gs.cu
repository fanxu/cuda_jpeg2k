#include "hip/hip_runtime.h"
/**
 * @file gs.cu
 *
 * @author Kamil Balwierz
 */

#include <math.h>
#include <time.h>
#include "gs.h"


int gram_schmidt(int N, type_data* output, type_data *dinput, type_data *eValues, int J, type_data er) {
	hipblasStatus_t status;
	int j, k;
	type_data *dT = 0;
	status = cublasAlloc(N*N, sizeof (dT[0]), (void**) &dT);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "! device memory allocation error (dT)\n");
		return EXIT_FAILURE;
	}
	type_data *doutput = 0;
	status = cublasAlloc(N*N, sizeof (doutput[0]), (void**) &doutput);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "! device memory allocation error (doutput)\n");
		return EXIT_FAILURE;
	}
	if (eValues == 0) {
		fprintf(stderr, "! host memory allocation error: T\n");
		return EXIT_FAILURE;
	}
	type_data *dU = 0;
	status = cublasAlloc(N, sizeof (dU[0]), (void**) &dU);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "! device memory allocation error (dU)\n");
		return EXIT_FAILURE;
	}

//	int iter = 0;
	type_data a;
	for (k = 0; k < N; k++) {
		hipblasScopy(N, &dinput[k * N], 1, &dT[k * N], 1);
		a = 0.0;
		for (j = 0; j < J; j++) {
			hipblasSgemv('t', N, N, 1.0, dinput, N, &dT[k * N], 1, 0.0, &doutput[k * N], 1);
			if (k > 0) {
				hipblasSgemv('t', N, k, 1.0, doutput, N, &doutput[k * N], 1, 0.0, dU, 1);
				hipblasSgemv('n', N, k, -1.0, doutput, N, dU, 1, 1.0, &doutput[k * N], 1);
			}
			hipblasSscal(N, 1.0 / hipblasSnrm2(N, &doutput[k * N], 1), &doutput[k * N], 1);
			hipblasSgemv('n', N, N, 1.0, dinput, N, &doutput[k * N], 1, 0.0, &dT[k * N], 1);
			if (k > 0) {
				hipblasSgemv('t', N, k, 1.0, dT, N, &dT[k * N], 1, 0.0, dU, 1);
				hipblasSgemv('n', N, k, -1.0, dT, N, dU, 1, 1.0, &dT[k * N], 1);
			}
			eValues[k] = hipblasSnrm2(N, &dT[k * N], 1);
			hipblasSscal(N, 1.0 / eValues[k], &dT[k * N], 1);
			if (fabs(a - eValues[k]) < er * eValues[k]) break;
			a = eValues[k];
//			iter++;
		}
//		printf("iter %d\n", iter);
//		iter = 0;
		hipblasSger(N, N, -eValues[k], &dT[k * N], 1, &doutput[k * N], 1, dinput, N);
	}
	for (k = 0; k < N; k++) {
		hipblasSscal(N, eValues[k], &dT[k * N], 1);
	}
//	hipblasSgemm('n', 'n', N, N, N, 1.0, dT, N, doutput, N, 0.0, output, N);
	hipblasGetMatrix(N, N, sizeof (doutput[0]), doutput, N, output, N);
	status = cublasFree(doutput);
	status = cublasFree(dT);
	return EXIT_SUCCESS;
}
