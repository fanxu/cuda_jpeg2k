#include "hip/hip_runtime.h"
/*
 * shift.c
 *
 *  Created on: Nov 30, 2011
 *      Author: miloszc
 */

#include <stdint.h>
#include "shift.h"

#define BLOCK_SIZE 16
#define TILE_SIZEX 32
#define TILE_SIZEY 32

void __global__ mean_shift_kernel(type_data *idata, const uint16_t width, const uint16_t height, const type_data mean) {
	int i = threadIdx.x;
	int j = threadIdx.y;
	int n = i + blockIdx.x * TILE_SIZEX;
	int m = j + blockIdx.y * TILE_SIZEY;
	int idx = n + m * width;

	while(j < TILE_SIZEY && m < height)
	{
		while(i < TILE_SIZEX && n < width)
		{
			idata[idx] = idata[idx] - mean;
			i += BLOCK_SIZE;
			n = i + blockIdx.x * TILE_SIZEX;
			idx = n + m * width;
		}
		i = threadIdx.x;
		j += BLOCK_SIZE;
		n = i + blockIdx.x * TILE_SIZEX;
		m = j + blockIdx.y * TILE_SIZEY;
		idx = n + m * width;
	}
}

void shit(type_data *idata, const uint16_t w, const uint16_t h, const type_data mean) {
	dim3 dimGrid((w + (TILE_SIZEX - 1))/TILE_SIZEX, (h + (TILE_SIZEY - 1))/TILE_SIZEY);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

	mean_shift_kernel<<<dimGrid, dimBlock>>>(idata, w, h, mean);
}
