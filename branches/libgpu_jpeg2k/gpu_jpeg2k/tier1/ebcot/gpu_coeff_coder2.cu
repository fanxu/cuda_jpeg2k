#include "hip/hip_runtime.h"
//#define DEBUG_MQ

#ifdef DEBUG_MQ
__device__ int l = 0;
__device__ int Cstates[1200000];
#endif

#include "gpu_coeff_coder2.cuh"
extern "C" {
	#include "../../misc/memory_management.cuh"
}
namespace GPU_JPEG2K
{
	#include "gpu_mq-coder.cuh"

__device__ void SetMaskedBits(unsigned int &reg, unsigned int mask, unsigned int bits)
{
	reg = (reg & ~mask) | (bits & mask);
}

__device__ void SetNthBit(unsigned int &reg, unsigned int n)
{
	SetMaskedBits(reg, 1 << n, 1 << n);
}

__device__ void ResetNthBit(unsigned int &reg, unsigned int n)
{
	SetMaskedBits(reg, 1 << n, 0);
}

typedef struct
{
	CoefficientState tl;
	CoefficientState t;
	CoefficientState tr;
	
	CoefficientState l;
	CoefficientState c;
	CoefficientState r;
	
	CoefficientState bl;
	CoefficientState b;
	CoefficientState br;

	short pos;
} CtxWindow;

__device__ void debug_print(float *val, int tid)
{
//	if(tid == 3)
//		printf("dist:%f\n", *val);
}


__device__ void down(CodeBlockAdditionalInfo &info, CtxWindow &window, CoefficientState *coeffs)
{
	window.tr = coeffs[window.pos + 1 - info.width];
	window.r = coeffs[window.pos + 1];
	window.br = coeffs[window.pos + 1 + info.width];
}

__device__ void up(CtxWindow &window, CoefficientState *coeffs)
{
	coeffs[window.pos - 1] = window.l;
}

__device__ void shift(CtxWindow &window)
{
	window.tl = window.t; window.t = window.tr; window.tr = 0; // top layer
	window.l = window.c; window.c = window.r; window.r = 0; // middle layer
	window.bl = window.b; window.b = window.br; window.br = 0; // bottom layer
	window.pos += 1;
}

typedef int CtxReg;

#define TRIMASK 0x249 //((1 << 0) | (1 << 3) | (1 << 6) | (1 << 9))

__device__ CtxReg buildCtxReg(CtxWindow &window, unsigned char bitoffset)
{
	CtxReg reg = 0;

	reg |= ((window.tl >> (bitoffset + 9)) & 1) << 0;
	reg |= ((window.t >> (bitoffset + 9)) & 1) << 1;
	reg |= ((window.tr >> (bitoffset + 9)) & 1) << 2;
	reg |= ((window.l >> (bitoffset)) & TRIMASK) << 3;
	reg |= ((window.c >> (bitoffset)) & TRIMASK) << 4;
	reg |= ((window.r >> (bitoffset)) & TRIMASK) << 5;
	reg |= ((window.bl >> (bitoffset)) & 1) << 15;
	reg |= ((window.b >> (bitoffset)) & 1) << 16;
	reg |= ((window.br >> (bitoffset)) & 1) << 17;
	
	return reg;
}

/*__device__ void updateWindow(CtxWindow &window, CtxReg reg, unsigned char bitoffset)
{
	window.tl = (window.tl & ~(1 << (bitoffset + 9)) | ((reg >> 0) & 1) << (bitoffset + 9);
	window.t = (window.t & ~(1 << (bitoffset + 9)) | ((reg >> 1) & 1) << (bitoffset + 9);
	window.tr = (window.tr & ~(1 << (bitoffset + 9)) | ((reg >> 2) & 1) << (bitoffset + 9);
	window.l = (window.l & ~(TRIMASK << (bitoffset)) | ((reg >> 3) & TRIMASK) << (bitoffset);
	window.c = (window.c & ~(TRIMASK << (bitoffset)) | ((reg >> 4) & TRIMASK) << (bitoffset);
	window.r = (window.r & ~(TRIMASK << (bitoffset)) | ((reg >> 5) & TRIMASK) << (bitoffset);
	window.bl = (window.bl & ~(1 << (bitoffset)) | ((reg >> 0) & 15) << (bitoffset);
	window.b = (window.b & ~(1 << (bitoffset)) | ((reg >> 0) & 16) << (bitoffset);
	window.br = (window.br & ~(1 << (bitoffset)) | ((reg >> 0) & 17) << (bitoffset);
}*/

	__constant__ unsigned char SPCXLUT[3][512] = {
		{
			0, 1, 3, 3, 1, 2, 3, 3, 5, 6, 7, 7, 6, 6, 7, 7, 0, 1, 3, 3,
			1, 2, 3, 3, 5, 6, 7, 7, 6, 6, 7, 7, 5, 6, 7, 7, 6, 6, 7, 7,
			8, 8, 8, 8, 8, 8, 8, 8, 5, 6, 7, 7, 6, 6, 7, 7, 8, 8, 8, 8,
			8, 8, 8, 8, 1, 2, 3, 3, 2, 2, 3, 3, 6, 6, 7, 7, 6, 6, 7, 7,
			1, 2, 3, 3, 2, 2, 3, 3, 6, 6, 7, 7, 6, 6, 7, 7, 6, 6, 7, 7,
			6, 6, 7, 7, 8, 8, 8, 8, 8, 8, 8, 8, 6, 6, 7, 7, 6, 6, 7, 7,
			8, 8, 8, 8, 8, 8, 8, 8, 3, 3, 4, 4, 3, 3, 4, 4, 7, 7, 7, 7,
			7, 7, 7, 7, 3, 3, 4, 4, 3, 3, 4, 4, 7, 7, 7, 7, 7, 7, 7, 7,
			7, 7, 7, 7, 7, 7, 7, 7, 8, 8, 8, 8, 8, 8, 8, 8, 7, 7, 7, 7,
			7, 7, 7, 7, 8, 8, 8, 8, 8, 8, 8, 8, 3, 3, 4, 4, 3, 3, 4, 4,
			7, 7, 7, 7, 7, 7, 7, 7, 3, 3, 4, 4, 3, 3, 4, 4, 7, 7, 7, 7,
			7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 8, 8, 8, 8, 8, 8, 8, 8,
			7, 7, 7, 7, 7, 7, 7, 7, 8, 8, 8, 8, 8, 8, 8, 8, 1, 2, 3, 3,
			2, 2, 3, 3, 6, 6, 7, 7, 6, 6, 7, 7, 1, 2, 3, 3, 2, 2, 3, 3,
			6, 6, 7, 7, 6, 6, 7, 7, 6, 6, 7, 7, 6, 6, 7, 7, 8, 8, 8, 8,
			8, 8, 8, 8, 6, 6, 7, 7, 6, 6, 7, 7, 8, 8, 8, 8, 8, 8, 8, 8,
			2, 2, 3, 3, 2, 2, 3, 3, 6, 6, 7, 7, 6, 6, 7, 7, 2, 2, 3, 3,
			2, 2, 3, 3, 6, 6, 7, 7, 6, 6, 7, 7, 6, 6, 7, 7, 6, 6, 7, 7,
			8, 8, 8, 8, 8, 8, 8, 8, 6, 6, 7, 7, 6, 6, 7, 7, 8, 8, 8, 8,
			8, 8, 8, 8, 3, 3, 4, 4, 3, 3, 4, 4, 7, 7, 7, 7, 7, 7, 7, 7,
			3, 3, 4, 4, 3, 3, 4, 4, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7, 7,
			7, 7, 7, 7, 8, 8, 8, 8, 8, 8, 8, 8, 7, 7, 7, 7, 7, 7, 7, 7,
			8, 8, 8, 8, 8, 8, 8, 8, 3, 3, 4, 4, 3, 3, 4, 4, 7, 7, 7, 7,
			7, 7, 7, 7, 3, 3, 4, 4, 3, 3, 4, 4, 7, 7, 7, 7, 7, 7, 7, 7,
			7, 7, 7, 7, 7, 7, 7, 7, 8, 8, 8, 8, 8, 8, 8, 8, 7, 7, 7, 7,
			7, 7, 7, 7, 8, 8, 8, 8, 8, 8, 8, 8
		},
		{
			0, 1, 5, 6, 1, 2, 6, 6, 3, 3, 7, 7, 3, 3, 7, 7, 0, 1, 5, 6,
			1, 2, 6, 6, 3, 3, 7, 7, 3, 3, 7, 7, 3, 3, 7, 7, 3, 3, 7, 7,
			4, 4, 7, 7, 4, 4, 7, 7, 3, 3, 7, 7, 3, 3, 7, 7, 4, 4, 7, 7,
			4, 4, 7, 7, 1, 2, 6, 6, 2, 2, 6, 6, 3, 3, 7, 7, 3, 3, 7, 7,
			1, 2, 6, 6, 2, 2, 6, 6, 3, 3, 7, 7, 3, 3, 7, 7, 3, 3, 7, 7,
			3, 3, 7, 7, 4, 4, 7, 7, 4, 4, 7, 7, 3, 3, 7, 7, 3, 3, 7, 7,
			4, 4, 7, 7, 4, 4, 7, 7, 5, 6, 8, 8, 6, 6, 8, 8, 7, 7, 8, 8,
			7, 7, 8, 8, 5, 6, 8, 8, 6, 6, 8, 8, 7, 7, 8, 8, 7, 7, 8, 8,
			7, 7, 8, 8, 7, 7, 8, 8, 7, 7, 8, 8, 7, 7, 8, 8, 7, 7, 8, 8,
			7, 7, 8, 8, 7, 7, 8, 8, 7, 7, 8, 8, 6, 6, 8, 8, 6, 6, 8, 8,
			7, 7, 8, 8, 7, 7, 8, 8, 6, 6, 8, 8, 6, 6, 8, 8, 7, 7, 8, 8,
			7, 7, 8, 8, 7, 7, 8, 8, 7, 7, 8, 8, 7, 7, 8, 8, 7, 7, 8, 8,
			7, 7, 8, 8, 7, 7, 8, 8, 7, 7, 8, 8, 7, 7, 8, 8, 1, 2, 6, 6,
			2, 2, 6, 6, 3, 3, 7, 7, 3, 3, 7, 7, 1, 2, 6, 6, 2, 2, 6, 6,
			3, 3, 7, 7, 3, 3, 7, 7, 3, 3, 7, 7, 3, 3, 7, 7, 4, 4, 7, 7,
			4, 4, 7, 7, 3, 3, 7, 7, 3, 3, 7, 7, 4, 4, 7, 7, 4, 4, 7, 7,
			2, 2, 6, 6, 2, 2, 6, 6, 3, 3, 7, 7, 3, 3, 7, 7, 2, 2, 6, 6,
			2, 2, 6, 6, 3, 3, 7, 7, 3, 3, 7, 7, 3, 3, 7, 7, 3, 3, 7, 7,
			4, 4, 7, 7, 4, 4, 7, 7, 3, 3, 7, 7, 3, 3, 7, 7, 4, 4, 7, 7,
			4, 4, 7, 7, 6, 6, 8, 8, 6, 6, 8, 8, 7, 7, 8, 8, 7, 7, 8, 8,
			6, 6, 8, 8, 6, 6, 8, 8, 7, 7, 8, 8, 7, 7, 8, 8, 7, 7, 8, 8,
			7, 7, 8, 8, 7, 7, 8, 8, 7, 7, 8, 8, 7, 7, 8, 8, 7, 7, 8, 8,
			7, 7, 8, 8, 7, 7, 8, 8, 6, 6, 8, 8, 6, 6, 8, 8, 7, 7, 8, 8,
			7, 7, 8, 8, 6, 6, 8, 8, 6, 6, 8, 8, 7, 7, 8, 8, 7, 7, 8, 8,
			7, 7, 8, 8, 7, 7, 8, 8, 7, 7, 8, 8, 7, 7, 8, 8, 7, 7, 8, 8,
			7, 7, 8, 8, 7, 7, 8, 8, 7, 7, 8, 8
		},
		{
			0, 3, 1, 4, 3, 6, 4, 7, 1, 4, 2, 5, 4, 7, 5, 7, 0, 3, 1, 4,
			3, 6, 4, 7, 1, 4, 2, 5, 4, 7, 5, 7, 1, 4, 2, 5, 4, 7, 5, 7,
			2, 5, 2, 5, 5, 7, 5, 7, 1, 4, 2, 5, 4, 7, 5, 7, 2, 5, 2, 5,
			5, 7, 5, 7, 3, 6, 4, 7, 6, 8, 7, 8, 4, 7, 5, 7, 7, 8, 7, 8,
			3, 6, 4, 7, 6, 8, 7, 8, 4, 7, 5, 7, 7, 8, 7, 8, 4, 7, 5, 7,
			7, 8, 7, 8, 5, 7, 5, 7, 7, 8, 7, 8, 4, 7, 5, 7, 7, 8, 7, 8,
			5, 7, 5, 7, 7, 8, 7, 8, 1, 4, 2, 5, 4, 7, 5, 7, 2, 5, 2, 5,
			5, 7, 5, 7, 1, 4, 2, 5, 4, 7, 5, 7, 2, 5, 2, 5, 5, 7, 5, 7,
			2, 5, 2, 5, 5, 7, 5, 7, 2, 5, 2, 5, 5, 7, 5, 7, 2, 5, 2, 5,
			5, 7, 5, 7, 2, 5, 2, 5, 5, 7, 5, 7, 4, 7, 5, 7, 7, 8, 7, 8,
			5, 7, 5, 7, 7, 8, 7, 8, 4, 7, 5, 7, 7, 8, 7, 8, 5, 7, 5, 7,
			7, 8, 7, 8, 5, 7, 5, 7, 7, 8, 7, 8, 5, 7, 5, 7, 7, 8, 7, 8,
			5, 7, 5, 7, 7, 8, 7, 8, 5, 7, 5, 7, 7, 8, 7, 8, 3, 6, 4, 7,
			6, 8, 7, 8, 4, 7, 5, 7, 7, 8, 7, 8, 3, 6, 4, 7, 6, 8, 7, 8,
			4, 7, 5, 7, 7, 8, 7, 8, 4, 7, 5, 7, 7, 8, 7, 8, 5, 7, 5, 7,
			7, 8, 7, 8, 4, 7, 5, 7, 7, 8, 7, 8, 5, 7, 5, 7, 7, 8, 7, 8,
			6, 8, 7, 8, 8, 8, 8, 8, 7, 8, 7, 8, 8, 8, 8, 8, 6, 8, 7, 8,
			8, 8, 8, 8, 7, 8, 7, 8, 8, 8, 8, 8, 7, 8, 7, 8, 8, 8, 8, 8,
			7, 8, 7, 8, 8, 8, 8, 8, 7, 8, 7, 8, 8, 8, 8, 8, 7, 8, 7, 8,
			8, 8, 8, 8, 4, 7, 5, 7, 7, 8, 7, 8, 5, 7, 5, 7, 7, 8, 7, 8,
			4, 7, 5, 7, 7, 8, 7, 8, 5, 7, 5, 7, 7, 8, 7, 8, 5, 7, 5, 7,
			7, 8, 7, 8, 5, 7, 5, 7, 7, 8, 7, 8, 5, 7, 5, 7, 7, 8, 7, 8,
			5, 7, 5, 7, 7, 8, 7, 8, 7, 8, 7, 8, 8, 8, 8, 8, 7, 8, 7, 8,
			8, 8, 8, 8, 7, 8, 7, 8, 8, 8, 8, 8, 7, 8, 7, 8, 8, 8, 8, 8,
			7, 8, 7, 8, 8, 8, 8, 8, 7, 8, 7, 8, 8, 8, 8, 8, 7, 8, 7, 8,
			8, 8, 8, 8, 7, 8, 7, 8, 8, 8, 8, 8
		}
	};

__device__ unsigned char getSPCX(CtxReg c, unsigned char i, unsigned char subband)
{
	return SPCXLUT[subband][(c >> (3 * i)) & 0x1FF];
}

	/* sign context in the following format
		index:
			first (MSB) bit V0 significance (1 significant, 0 insignificant)
			second bit V0 sign (0 positive, 1 negative)

			next 2 bits same for H0
			next 2 bits same for H1
			next 2 bits same for V1
			
		value:
			the response contains two pieces of information
			1. context label on the 4 least significant bits
			2. XORbit on the 5-th bit from the end (5-th least significant bit)
	*/

	__constant__ unsigned char signcxlut[256] = {
		 9,  9, 10, 26,  9,  9, 10, 26, 12, 12, 13, 11, 28, 28, 27, 29,  9,  9, 10, 26,
		 9,  9, 10, 26, 12, 12, 13, 11, 28, 28, 27, 29, 12, 12, 13, 11, 12, 12, 13, 11,
		12, 12, 13, 11,  9,  9, 10, 26, 28, 28, 27, 29, 28, 28, 27, 29,  9,  9, 10, 26,
		28, 28, 27, 29,  9,  9, 10, 26,  9,  9, 10, 26, 12, 12, 13, 11, 28, 28, 27, 29,
		 9,  9, 10, 26,  9,  9, 10, 26, 12, 12, 13, 11, 28, 28, 27, 29, 12, 12, 13, 11,
		12, 12, 13, 11, 12, 12, 13, 11,  9,  9, 10, 26, 28, 28, 27, 29, 28, 28, 27, 29,
		 9,  9, 10, 26, 28, 28, 27, 29, 10, 10, 10,  9, 10, 10, 10,  9, 13, 13, 13, 12,
		27, 27, 27, 28, 10, 10, 10,  9, 10, 10, 10,  9, 13, 13, 13, 12, 27, 27, 27, 28,
		13, 13, 13, 12, 13, 13, 13, 12, 13, 13, 13, 12, 10, 10, 10,  9, 27, 27, 27, 28,
		27, 27, 27, 28, 10, 10, 10,  9, 27, 27, 27, 28, 26, 26,  9, 26, 26, 26,  9, 26,
		11, 11, 12, 11, 29, 29, 28, 29, 26, 26,  9, 26, 26, 26,  9, 26, 11, 11, 12, 11,
		29, 29, 28, 29, 11, 11, 12, 11, 11, 11, 12, 11, 11, 11, 12, 11, 26, 26,  9, 26,
		29, 29, 28, 29, 29, 29, 28, 29, 26, 26,  9, 26, 29, 29, 28, 29
	};

__device__ unsigned char getSICX(CtxReg sig, CtxReg sign, unsigned char i)
{
	return signcxlut[
			((sig >> (i * 3)) & 0xAA) |
			(((sign >> (i * 3)) & 0xAA) >> 1)
		];
}

__device__ unsigned char getMRCX(CtxReg sig, CoefficientState local, unsigned char i)
{
	if((local >> (12 + 3 * i)) & 1)
		return 16;
	else if(((sig >> (3 * i)) & 0x1EF) == 0)
		return 14;
	else
		return 15;
}

template<class T> __device__ T min(T val1, T val2)
{
	if(val1 > val2)
	{
		return val2;
	} else
	{
		return val1;
	}
}

template<class T> __device__ T max(T val1, T val2)
{
	if(val1 < val2)
	{
		return val2;
	} else
	{
		return val1;
	}
}

__constant__ float distWeights[2][4][4] = {
{//Lossless
//		LH,      HL,      HH,     LLend
	{0.1000f, 0.1000f, 0.0500f, 1.0000f},  //level 0 = biggest subbands (unimportant)
	{0.2000f, 0.2000f, 0.1000f, 1.0000f},  //      1
	{0.4000f, 0.4000f, 0.2000f, 1.0000f},  //      2
	{0.8000f, 0.8000f, 0.4000f, 1.0000f}   //      3 = smallest, contains LL
}, {//Lossy
/*	{ 0.0010f, 0.0010f, 0.0005f, 1.0000f},
	{ 0.1000f, 0.1000f, 0.0250f, 1.0000f},
	{ 0.3000f, 0.3000f, 0.0800f, 1.0000f},
	{ 0.8000f, 0.8000f, 0.4000f, 1.0000f}*/
	{0.0100f, 0.0100f, 0.0050f, 1.0000f},
	{0.2000f, 0.2000f, 0.1000f, 1.0000f},
	{0.4000f, 0.4000f, 0.2000f, 1.0000f},
	{0.8000f, 0.8000f, 0.4000f, 1.0000f}
} };

__device__ float getDISW(CodeBlockAdditionalInfo *info)
{
	return distWeights[info->compType][min<byte>(info->dwtLevel, 3)][info->subband] * info->stepSize * info->stepSize / ((float)(info->width * info->height));
}

/*__device__ void binary_printf(unsigned int in)
{
	for(int i = 0; i < 32; i++) {
		if((in >> (31 - i)) & 1)
			printf("1");
		else
			printf("0");
		if(i % 8 == 7)
			printf(" ");
	}

	printf("\n");
}*/

class RLEncodeFunctor {
public:
	__device__ char operator()(CtxWindow window, MQEncoder &enc, CXD &cxd_pair)
	{
		char rest = 0;

		if((window.c & TRIMASK) == 0)
		{
			save_cxd_pair(cxd_pair, 0, CX_UNI);
			rest = -2;
		}
		else
		{
			while(rest < 4 && ((window.c >> (3 * rest)) & 1) == 0)
				rest++;
			
			save_cxd_pair(cxd_pair, 1, CX_UNI);
			save_cxd_pair(cxd_pair, rest >> 1, CX_RUN);
			save_cxd_pair(cxd_pair, rest & 1, CX_RUN);
		}

		return rest;
	}
};

class RLDecodeFunctor {
public:
	__device__ char operator()(CtxWindow &window, MQDecoder &dec, CXD &cxd_pair)
	{
		char rest = 0;

		if(mqDecode(dec, CX_RUN) == 0)
		{
			rest = -2;
		}
		else
		{
			rest = mqDecode(dec, CX_UNI) & 1;
			rest <<= 1;
			rest |= mqDecode(dec, CX_UNI) & 1;

			window.c |= 1 << (3 * rest);
		}

		return rest;
	}
};

class SigEncodeFunctor {
public:
	__device__ void operator()(CtxWindow &window, CtxReg &sig, MQEncoder &enc, CXD &cxd_pair, int stripId, int subband)
	{
		save_cxd_pair(cxd_pair, (window.c >> (3 * stripId)) & 1, getSPCX(sig, stripId, subband));
	}
};

class SigDecodeFunctor {
public:
	__device__ void operator()(CtxWindow &window, CtxReg sig, MQDecoder &dec, CXD &cxd_pair, int stripId, int subband)
	{
		window.c |= mqDecode(dec, getSPCX(sig, stripId, subband)) << (3 * stripId);
	}
};

class SignEncodeFunctor
{
public:
	__device__ void operator()(CtxWindow &window, CtxReg &sig, MQEncoder &enc, CXD &cxd_pair, int stripId)
	{
		unsigned char cx = getSICX(sig, buildCtxReg(window, 13), stripId);
		save_cxd_pair(cxd_pair, (short) (((window.c >> (13 + 3 * stripId)) & 1) ^ ((cx >> 4) & 1)), cx & 0xF);
	}
};

class SignDecodeFunctor
{
public:
	__device__ void operator()(CtxWindow &window, CtxReg sig, MQDecoder &dec, CXD &cxd_pair, int stripId)
	{
		unsigned char cx = getSICX(sig, buildCtxReg(window, 13), stripId);

		window.c |= (mqDecode(dec, cx & 0xF) ^ ((cx >> 4) & 1) & 1) << (13 + 3 * stripId);
	}	
};

template <class RLCodingFunctor, class SigCodingFunctor, class SignCodingFunctor, typename MQCoderStateType>
class CleanUpPassFunctor
{
public:
	__device__ void operator()(const CodeBlockAdditionalInfo &info, CtxWindow &window, MQCoderStateType &mq, CXD &cxd_pair, float *sum_dist, unsigned char bitplane)
	{
		char rest;

		CtxReg sig = buildCtxReg(window, 1); // significance context

		rest = -1;
		if((window.c & (TRIMASK << 14)) == 0 && sig == 0) // all contexts in stripe are equal to zero
		{
			rest = RLCodingFunctor()(window, mq, cxd_pair);
			if(rest == -2)
				return;
		}

		for(int k = 0; k < 4; k++)
		{
			if(/*	((window.c >> ( 1 + 3 * k)) & 1) == 0 &&   // check if coefficient is non-significant (sigma)
				((window.c >> ( 2 + 3 * k)) & 1) == 0 &&   // check if coefficient hasn't been coded already (pi)
				((window.c >> (14 + 3 * k)) & 1) == 0)    // forbidden state indicating out of bounds (late sigma)*/
				((window.c >> (3 * k)) & 0x4006) == 0)
			{
				if(rest >= 0)
					rest--;
				else
					SigCodingFunctor()(window, sig, mq, cxd_pair, k, info.subband);
			
				if((window.c >> (3 * k)) & 1) // check if magnitude is 1
				{
					*sum_dist -= (float)((1<<bitplane)*(1<<bitplane));
//					debug_print(sum_dist, threadIdx.x);
//					if(blockIdx.x * blockDim.x + threadIdx.x == 0)
//					printf("clu:%f tid:%d\n", *sum_dist, blockIdx.x * blockDim.x + threadIdx.x);
					SetNthBit(window.c, 1 + 3 * k); // set k-th significant state
					sig = buildCtxReg(window, 1); // rebuild significance register

					SignCodingFunctor()(window, sig, mq, cxd_pair, k);
				}
			}
		}
	}
};

template <class SigCodingFunctor, class SignCodingFunctor, typename MQCoderStateType>
class SigPropPassFunctor {
public:
__device__ void operator()(const CodeBlockAdditionalInfo &info, CtxWindow &window, MQCoderStateType &mq, CXD &cxd_pair, float *sum_dist, unsigned char bitplane)
{
	CtxReg sig = buildCtxReg(window, 1); // build significance context register

	for(int i = 0; i < 4; i++)
	{
		// not significant with non-zero context
		if(/*	((window.c >> (1 + 3 * i)) & 1) == 0 &&
			((window.c >> (14 + 3 * i)) & 1) == 0 && // out of bounds
			getSPCX(sig, i, subband) > 0)*/
			(((window.c >> (3 * i)) & 0x4002) == 0) &&
			((sig >> (3 * i)) & 0x1EF) != 0)
		{
			SigCodingFunctor()(window, sig, mq, cxd_pair, i, info.subband);

			// if magnitude bit is one
			if((window.c >> (3 * i)) & 1)
			{
				*sum_dist -= (float)((1<<bitplane)*(1<<bitplane));
//				debug_print(sum_dist, threadIdx.x);
//				if(blockIdx.x * blockDim.x + threadIdx.x == 0)
//				printf("sig:%f tid:%d\n", *sum_dist, blockIdx.x * blockDim.x + threadIdx.x);
				SetNthBit(window.c, 1 + (3 * i));
				sig = buildCtxReg(window, 1); // rebuild

				SignCodingFunctor()(window, sig, mq, cxd_pair, i);
			}

			// set pi (already coded)
			SetNthBit(window.c, 2 + (3 * i));
		}
		else
			// unset pi (already coded)
			ResetNthBit(window.c, 2 + (3 * i));
	}
}
};

class MagRefEncodeFunctor {
public:
	__device__ void operator()(MQEncoder &enc, CXD &cxd_pair, CtxWindow &window, int stripId)
	{
		save_cxd_pair(cxd_pair, (window.c >> (3 * stripId)) & 1, getMRCX(buildCtxReg(window, 1), window.c, stripId));
	}
};

class MagRefDecodeFunctor {
public:
	__device__ void operator()(MQDecoder &dec, CXD &cxd_pair, CtxWindow &window, int stripId)
	{
		window.c |= (mqDecode(dec, getMRCX(buildCtxReg(window, 1), window.c, stripId)) << (3 * stripId));
	}
};

template <class MagRefCodingFunctor, typename MQCoderStateType>
class MagRefPassFunctor {
public:
__device__ void operator()(const CodeBlockAdditionalInfo &info, CtxWindow &window, MQCoderStateType &mq, CXD &cxd_pair, float *sum_dist, unsigned char bitplane)
{
	for(int i = 0; i < 4; i++)
	{
		if(//csSignificant(st) && !csAlreadyCoded(st) && not out of bounds
			((window.c >> (3 * i)) & 0x4006) == 0x2)
		{
			*sum_dist -= (float)((1<<bitplane)*(1<<bitplane));
//			debug_print(sum_dist, threadIdx.x);
//			if(blockIdx.x * blockDim.x + threadIdx.x == 0)
//			printf("mgr:%f tid:%d\n", *sum_dist, blockIdx.x * blockDim.x + threadIdx.x);
			MagRefCodingFunctor()(mq, cxd_pair, window, i);
			SetNthBit(window.c, 3 * i + 12);
		}
	}
}
};

__device__ void initCoeffs(const CodeBlockAdditionalInfo &info, CoefficientState *coeffs)
{
	unsigned char signOffset = sizeof(int) * 8 - 1;

	for(int i = 0; i < info.width; i++)
		for(int j = 0; j < info.stripeNo; j++)
		{
			CoefficientState st = 0;
			int c;

			for(int k = 0; k < 4; k++)
				if(4 * j + k < info.height)
				{
					c = info.coefficients[(4 * j + k) * info.nominalWidth + i];
//					binary_printf(c);
					//Cstates[l++] = (4 * j + k) * info.nominalWidth + i;
					st |= (((c >> signOffset) & 1) << (13 + 3 * k));
				}
				else
					st |= (1 << (14 + 3 * k));

			coeffs[j * info.width + i] = st;
		}
}

__device__ void initDecodingCoeffs(const CodeBlockAdditionalInfo &info, CoefficientState *coeffs)
{
	for(int i = 0; i < info.width; i++)
		for(int j = 0; j < info.stripeNo; j++)
		{
			CoefficientState st = 0;

			for(int k = 0; k < 4; k++)
				if(4 * j + k < info.height)
					info.coefficients[(4 * j + k) * info.nominalWidth + i] = 0;
				else
					st |= (1 << (14 + 3 * k));

			coeffs[j * info.width + i] = st;
		}
}

__device__ void uploadSigns(const CodeBlockAdditionalInfo &info, CoefficientState *coeffs)
{
	unsigned char signOffset = sizeof(int) * 8 - 1;

	for(int i = 0; i < info.width; i++)
		for(int j = 0; j < info.stripeNo; j++)
		{
			CoefficientState st = coeffs[j * info.width + i];

			for(int k = 0; k < 4; k++)
				if(((st >> (14 + 3 * k)) & 1) == 0)
					info.coefficients[(4 * j + k) * info.nominalWidth + i] |= (((st >> (13 + 3 * k)) & 1) << signOffset);

			coeffs[j * info.width + i] = st;
		}
}

__device__ void fillMags(const CodeBlockAdditionalInfo &info, CoefficientState *coeffs, int bitplane)
{
	for(int i = 0; i < info.width; i++)
		for(int j = 0; j < info.stripeNo; j++)
		{
			CoefficientState st = coeffs[j * info.width + i];

			// clear magnitudes and already coded flags
			st &= ~(TRIMASK | (TRIMASK << 2));
			//st |= ((st & (TRIMASK << 1)) << 11);

			for(int k = 0; k < 4; k++)
				if(((st >> (14 + 3 * k)) & 1) == 0)
					st |= ((info.coefficients[(4 * j + k) * info.nominalWidth + i] >> bitplane) & 1) << (3 * k);

			coeffs[j * info.width + i] = st;
//			binary_printf(st);
		}
}

__device__ void uploadMags(const CodeBlockAdditionalInfo &info, CoefficientState *coeffs, int bitplane)
{
	for(int i = 0; i < info.width; i++)
		for(int j = 0; j < info.stripeNo; j++)
		{
			CoefficientState st = coeffs[j * info.width + i];

			for(int k = 0; k < 4; k++)
				if(((st >> (14 + 3 * k)) & 1) == 0)
					info.coefficients[(4 * j + k) * info.nominalWidth + i] |= (((st >> (3 * k)) & 1) << bitplane);

			// clear magnitudes and already coded flags
			st &= ~(TRIMASK | (TRIMASK << 2));

			coeffs[j * info.width + i] = st;
		}
}

__device__ void clearWindow(CtxWindow &w)
{
	w.bl = 0;
	w.b = 0;
	w.br = 0;

	w.l = 0;
	w.c = 0;
	w.r = 0;

	w.tl = 0;
	w.t = 0;
	w.tr = 0;
}

template <class PassFunctor, typename MQCoderStateType>
__device__ void BITPLANE_WINDOW_SCAN(CodeBlockAdditionalInfo &info, CoefficientState *coeffs, MQCoderStateType &enc, CXD &cxd_pair, float *sum_dist, unsigned char bitplane) {
	CtxWindow window;

	window.pos = -1;

	for(int j = 0; j < info.stripeNo; j++)
	{
		clearWindow(window);
		down(info, window, coeffs);
		shift(window);
		down(info, window, coeffs);
	
		PassFunctor()(info, window, enc, cxd_pair, sum_dist, bitplane);

		for(int k = 0; k < info.width - 2; k++)
		{
			shift(window);
			down(info, window, coeffs);
			PassFunctor()(info, window, enc, cxd_pair, sum_dist, bitplane);
			up(window, coeffs);
		}

		shift(window);
		PassFunctor()(info, window, enc, cxd_pair, sum_dist, bitplane);
		up(window, coeffs);
		shift(window);
		up(window, coeffs);

		window.pos--;
	}
}

class PCRD_EmptyFunctor
{
public:
	__device__ void operator()(MQEncoder state, CXD cxd_pair, MQEncoder *states, CXD *cxds, unsigned char &stateId, float sum_dist, PcrdCodeblock *pcrdCodeblock)
	{
	}
};

class PCRD_CollectMQStatesFunctor
{
public:
	__device__ void operator()(MQEncoder state, CXD cxd_pair, MQEncoder *states, CXD *cxds, unsigned char &stateId, float sum_dist, PcrdCodeblock *pcrdCodeblock)
	{
		cxds[stateId] = cxd_pair;
		//states[stateId++] = state;
		pcrdCodeblock[stateId++].dist = sum_dist;
	}
};

class CollectMQStatesFunctor
{
public:
	__device__ void operator()(MQEncoder state, CXD cxd_pair, MQEncoder *states, CXD *cxds, unsigned char &stateId, float sum_dist, PcrdCodeblock *pcrdCodeblock)
	{
		cxds[stateId] = cxd_pair;
		//states[stateId++] = state;
	}
};

template <class PostPassFunctor, class PostCodingFunctor>
__device__ void encode(CoefficientState *coeffs, byte *cxd_pairs, CodeBlockAdditionalInfo &info, MQEncoder *states, CXD *cxds, PcrdCodeblock *pcrdCodeblock = NULL)
{
	unsigned char leastSignificantBP = 31 - info.magbits;

	info.significantBits = 0;
	int c;
	for(int i = 0; i < info.width; i++)
		for(int j = 0; j < info.height; j++)
		{
			c = info.coefficients[j * info.nominalWidth + i];
//			binary_printf(c);
			int k;
			for(k = 30; k >= leastSignificantBP; k--)
				if((c >> k) & 1)
					break;

			if(k - leastSignificantBP + 1 > info.significantBits)
				info.significantBits = k - leastSignificantBP + 1;

		}

	CXD cxd_pair;
	cxdPairInit(cxd_pair, cxd_pairs);
	MQEncoder mqenc;
	//mqInitEnc(mqenc, out, cxd_pairs);
				
	unsigned char sid = 0;
	float sum_dist = 0.0f;

	if(pcrdCodeblock != NULL)
		pcrdCodeblock[sid].dist = 0;

	if(info.significantBits > 0)
	{
		//mqResetEnc(mqenc);
		
		initCoeffs(info, coeffs);
		
		// first plane
		fillMags(info, coeffs, leastSignificantBP + info.significantBits - 1);
		
//		printf("bitplane:%d tid:%d\n", leastSignificantBP + info.significantBits - 1, blockIdx.x * blockDim.x + threadIdx.x);

		BITPLANE_WINDOW_SCAN
		<CleanUpPassFunctor<RLEncodeFunctor, SigEncodeFunctor, SignEncodeFunctor, MQEncoder>, MQEncoder >
			(info, coeffs, mqenc, cxd_pair, &sum_dist, info.significantBits - 1);

		PostPassFunctor()(mqenc, cxd_pair, NULL/*states*/, cxds, sid, sum_dist, pcrdCodeblock);
		
		for(unsigned char i = 1; i < info.significantBits; i++)
		{
			fillMags(info, coeffs, leastSignificantBP + info.significantBits - i - 1);

//			printf("bitplane:%d tid:%d\n", leastSignificantBP + info.significantBits - i - 1, blockIdx.x * blockDim.x + threadIdx.x);
			
			BITPLANE_WINDOW_SCAN
			<SigPropPassFunctor<SigEncodeFunctor, SignEncodeFunctor, MQEncoder>, MQEncoder >
				(info, coeffs, mqenc, cxd_pair, &sum_dist, info.significantBits - i - 1);

			PostPassFunctor()(mqenc, cxd_pair, NULL/*states*/, cxds, sid, sum_dist, pcrdCodeblock);

			BITPLANE_WINDOW_SCAN
			<MagRefPassFunctor<MagRefEncodeFunctor, MQEncoder>, MQEncoder >
				(info, coeffs, mqenc, cxd_pair, &sum_dist, info.significantBits - i - 1);

			PostPassFunctor()(mqenc, cxd_pair, NULL/*states*/, cxds, sid, sum_dist, pcrdCodeblock);

			BITPLANE_WINDOW_SCAN
			<CleanUpPassFunctor<RLEncodeFunctor, SigEncodeFunctor, SignEncodeFunctor, MQEncoder>, MQEncoder >
				(info, coeffs, mqenc, cxd_pair, &sum_dist, info.significantBits - i - 1);

			PostPassFunctor()(mqenc, cxd_pair, NULL/*states*/, cxds, sid, sum_dist, pcrdCodeblock);
		}

		PostCodingFunctor()(mqenc, cxd_pair, NULL/*states*/, cxds, sid, sum_dist, pcrdCodeblock);
		//mqFlush(mqenc);
	}
}

__device__ void decode(CoefficientState *coeffs, CodeBlockAdditionalInfo &info, byte *in)
{
	CXD cxd_pair;
	MQDecoder mqdec;
	mqInitDec(mqdec, in, info.length);

	float sum_dist = 0.0f;

	if(info.significantBits > 0)
	{
		mqResetDec(mqdec);

		initDecodingCoeffs(info, coeffs);

		BITPLANE_WINDOW_SCAN
		<CleanUpPassFunctor<RLDecodeFunctor, SigDecodeFunctor, SignDecodeFunctor, MQDecoder>, MQDecoder>
			(info, coeffs, mqdec, cxd_pair, &sum_dist, 0);

		uploadMags(info, coeffs, 30 - info.magbits + info.significantBits);

		for(unsigned char i = 1; i < info.significantBits; i++)
		{
			BITPLANE_WINDOW_SCAN
			<SigPropPassFunctor<SigDecodeFunctor, SignDecodeFunctor, MQDecoder>, MQDecoder>
				(info, coeffs, mqdec, cxd_pair, &sum_dist, 0);

			BITPLANE_WINDOW_SCAN
			<MagRefPassFunctor<MagRefDecodeFunctor, MQDecoder>, MQDecoder>
				(info, coeffs, mqdec, cxd_pair, &sum_dist, 0);

			BITPLANE_WINDOW_SCAN
			<CleanUpPassFunctor<RLDecodeFunctor, SigDecodeFunctor, SignDecodeFunctor, MQDecoder>, MQDecoder>
				(info, coeffs, mqdec, cxd_pair, &sum_dist, 0);

			uploadMags(info, coeffs, 30 - info.magbits - i + info.significantBits);
		}

		uploadSigns(info, coeffs);
		//mqDecode(mqdec, CX_UNI);
	}
	else
	{
		for(int i = 0; i < info.height; i++)
			for(int j = 0; j < info.width; j++)
				info.coefficients[i * info.nominalWidth + j] = 0;
	}
}

__global__ void g_encode(CoefficientState *coeffBuffors, byte *cxd_pairs, int maxThreadBufforLength, CodeBlockAdditionalInfo *infos, int codeBlocks, MQEncoder *mqstates, CXD *cxds)
{
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;

	if(threadId >= codeBlocks)
		return;

	CodeBlockAdditionalInfo info = infos[threadId];

	info.length = 0;
	encode<PCRD_EmptyFunctor, CollectMQStatesFunctor>(coeffBuffors + info.magconOffset, cxd_pairs + threadId * maxThreadBufforLength, info, mqstates + threadId, cxds + threadId);

	infos[threadId].significantBits = info.significantBits;
	infos[threadId].length = info.length;
}

__global__ void g_encode_pcrd(CoefficientState *coeffBuffors, int maxThreadBufforLength, CodeBlockAdditionalInfo *infos, int codeBlocks, MQEncoder *mqstates, int maxStatesPerCodeblock, PcrdCodeblock *pcrdCodeblocks)
{
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;

	if(threadId >= codeBlocks)
		return;

	CodeBlockAdditionalInfo info = infos[threadId];

	info.length = 0;
	encode<PCRD_CollectMQStatesFunctor, PCRD_EmptyFunctor>(coeffBuffors + info.magconOffset, NULL, info, mqstates + threadId * maxStatesPerCodeblock, NULL, pcrdCodeblocks + threadId * maxStatesPerCodeblock);

	infos[threadId].significantBits = info.significantBits;
	infos[threadId].length = info.length;
}

__global__ void g_lengthCalculation(CodeBlockAdditionalInfo *infos, int codeBlocks, MQEncoder *mqstates, CXD *cxds)
{
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;

	if(threadId >= codeBlocks)
		return;

	if(infos[threadId].significantBits > 0) {
		/*infos[threadId].length = *//*mqFullFlush(mqstates[threadId]);*/
		infos[threadId].length = get_cxd_pairs_count(cxds[threadId]/*mqstates[threadId]*/);
		infos[threadId].codingPasses = infos[threadId].significantBits * 3 -2;
	}
	else {
		infos[threadId].length = 0;
		infos[threadId].codingPasses = 1;
	}
}

__global__ void g_lengthCalculation_pcrd(CodeBlockAdditionalInfo *infos, int codeBlocks, MQEncoder *mqstates, int maxStatesPerCodeBlock, PcrdCodeblock *pcrdCodeblocks, PcrdCodeblockInfo *pcrdCodeblockInfos)
{
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;

	if(threadId >= codeBlocks)
		return;

	mqstates += maxStatesPerCodeBlock * threadId;
	pcrdCodeblocks += maxStatesPerCodeBlock * threadId;
	pcrdCodeblockInfos += threadId;
	infos += threadId;

//	printf("%c[%d;%d;%dm", 27, 1, threadId * 2, 40);

	if(infos->significantBits > 0)
	{
		int nstates = (infos->significantBits - 1) * 3 + 1;

		int len;

//		pcrdCodeblockInfos->nStates = nstates;
		infos->codingPasses = nstates;

		pcrdCodeblocks[0].dist = 0;
		pcrdCodeblocks[0].L = 0;

		for(int i = 0; i < nstates; i++)
		{
			len = max<int>(mqFullFlush(mqstates[i]), 0);
//			mqstates[i].L = len;
			pcrdCodeblocks[i + 1].L = len;
			/*if(pcrdCodeblocks[i].L < 0)
			{
				printf("Error: Len < 0!\n");
			}*/
/*			if(threadId == 1)
			{
//				printf("L[%2d]:%6d %6f\n", i, pcrdCodeblocks[i].L, pcrdCodeblocks[i].dist);
			}*/
			pcrdCodeblocks[i + 1].dist *= ((len == 0) ? 0 : getDISW(infos));
//			pcrdCodeblocks[i + 1].dist *= getDISW(infos);
/*//			if(threadId == 1)
			{
//				printf("%f\n", getDISW(info));
//				printf("%d %f %d tid:%d\n", pcrdCodeblocks[i].L, pcrdCodeblocks[i].dist, pcrdCodeblockInfos->nStates, threadId);
				printf("L[%2d]:%6d %6f\n", i, pcrdCodeblocks[i].L, pcrdCodeblocks[i].dist);
			}*/
		}

		infos->length = len;
	}
	else
	{
//		printf("No significant bits!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
		mqstates[0].L = 0;
		pcrdCodeblocks[0].L = 0;
		infos->length = 0;
		infos->codingPasses = 1;
	}
}



__global__ void g_decode(CoefficientState *coeffBuffors, byte *inbuf, int maxThreadBufforLength, CodeBlockAdditionalInfo *infos, int codeBlocks)
{
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;

	if(threadId >= codeBlocks)
		return;

	CodeBlockAdditionalInfo info = infos[threadId];

	decode(coeffBuffors + info.magconOffset, info, inbuf + threadId * maxThreadBufforLength);
}

#include <stdio.h>

void launch_encode(dim3 gridDim, dim3 blockDim, CoefficientState *coeffBuffors, byte *cxd_pairs, int maxThreadBufforLength, CodeBlockAdditionalInfo *infos, int codeBlocks, mem_mg_t *mem_mg)
{
	// Initialize CUDA
	hipError_t cuerr;

	CXD *cxds = (CXD *)mem_mg->alloc->dev(sizeof(CXD) * codeBlocks, mem_mg->ctx);
	MQEncoder *mqstates = (MQEncoder *)mem_mg->alloc->dev(sizeof(MQEncoder) * codeBlocks, mem_mg->ctx);

//	printf("grid %d %d %d\nblock %d %d %d\n", gridDim.x, gridDim.y, gridDim.z, blockDim.x, blockDim.y, blockDim.z);

	g_encode<<<gridDim, blockDim>>>(coeffBuffors, cxd_pairs, maxThreadBufforLength, infos, codeBlocks, mqstates, cxds);
	hipDeviceSynchronize();
	if (cuerr = hipGetLastError()) {
		printf("g_encode error: %s\n", hipGetErrorString(cuerr));
		return;
	}

	g_lengthCalculation<<<(int) ceil(codeBlocks / 512.0f), 512>>>(infos, codeBlocks, mqstates, cxds);
	hipDeviceSynchronize();
	if (cuerr = hipGetLastError()) {
		printf("g_lengthCalculation error: %s\n", hipGetErrorString(cuerr));
		return;
	}

	mem_mg->dealloc->dev(mqstates, mem_mg->ctx);
	mem_mg->dealloc->dev(cxds, mem_mg->ctx);
}

void _launch_encode_pcrd(dim3 gridDim, dim3 blockDim, CoefficientState *coeffBuffors, int maxThreadBufforLength, CodeBlockAdditionalInfo *infos, int codeBlocks, const int maxMQStatesPerCodeBlock, PcrdCodeblock *pcrdCodeblocks, PcrdCodeblockInfo *pcrdCodeblockInfos, mem_mg_t *mem_mg) {
	MQEncoder *mqstates;
	mqstates = (MQEncoder *)mem_mg->alloc->dev(sizeof(MQEncoder) * codeBlocks * maxMQStatesPerCodeBlock, mem_mg->ctx);

	g_encode_pcrd<<<gridDim, blockDim>>>(coeffBuffors, maxThreadBufforLength, infos, codeBlocks, mqstates, maxMQStatesPerCodeBlock, pcrdCodeblocks);

	g_lengthCalculation_pcrd<<<(int) ceil(codeBlocks / 512.0f), 512>>>(infos, codeBlocks, mqstates, maxMQStatesPerCodeBlock, pcrdCodeblocks, pcrdCodeblockInfos);

	mem_mg->dealloc->dev(mqstates, mem_mg->ctx);
}


void launch_decode(dim3 gridDim, dim3 blockDim, CoefficientState *coeffBuffors, byte *inbuf, int maxThreadBufforLength, CodeBlockAdditionalInfo *infos, int codeBlocks)
{
	g_decode<<<gridDim, blockDim>>>(coeffBuffors, inbuf, maxThreadBufforLength, infos, codeBlocks);
}

}
